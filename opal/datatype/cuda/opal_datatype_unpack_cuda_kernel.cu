#include "hip/hip_runtime.h"
#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h> 


__global__ void opal_generic_simple_unpack_cuda_iov_kernel( ddt_cuda_iov_dist_t* cuda_iov_dist, int nb_blocks_used, unsigned char* source_base, unsigned char* destination_base)
{
    uint32_t i, _copy_count;
    size_t src_offset, dst_offset;
    uint8_t alignment;
    unsigned char *_source_tmp, *_destination_tmp;
    
    __shared__ uint32_t nb_tasks;
    
    if (threadIdx.x == 0) {
        nb_tasks = nb_blocks_used / gridDim.x;
        if (blockIdx.x < nb_blocks_used % gridDim.x) {
            nb_tasks ++;
        }
    }
    __syncthreads();
    
    for (i = 0; i < nb_tasks; i++) {
        src_offset = cuda_iov_dist[blockIdx.x + i * gridDim.x].src_offset;
        dst_offset = cuda_iov_dist[blockIdx.x + i * gridDim.x].dst_offset;
        _copy_count = cuda_iov_dist[blockIdx.x + i * gridDim.x].nb_elements;
        alignment = cuda_iov_dist[blockIdx.x + i * gridDim.x].element_alignment;
        
        if (threadIdx.x < _copy_count) {
            _source_tmp = source_base + src_offset + threadIdx.x * alignment;
            _destination_tmp = destination_base + dst_offset + threadIdx.x * alignment;
#if !defined (OPAL_DATATYPE_CUDA_DRY_RUN)
                if (alignment == ALIGNMENT_DOUBLE) {
                    *((long *)_destination_tmp) = *((long *)_source_tmp);
                } else if (alignment == ALIGNMENT_FLOAT) {
                    *((int *)_destination_tmp) = *((int *)_source_tmp);
                } else {
                    * _destination_tmp = *_source_tmp;
                }
        //   printf("src %p, %1.f | dst %p, %1.f\n", _source_tmp, *_source_tmp, _destination_tmp, *_destination_tmp);
#endif /* ! OPAL_DATATYPE_CUDA_DRY_RUN */
        }
    }
}
__global__ void unpack_contiguous_loop_cuda_kernel_global( uint32_t copy_loops,
                                                           size_t size,
                                                           OPAL_PTRDIFF_TYPE extent,
                                                           unsigned char* source,
                                                           unsigned char* destination )
{
    uint32_t _i, tid, num_threads;
    uint32_t gap, nb_elements;
    double *_source_tmp, *_destination_tmp, *_dst_disp_tmp;;
    
    tid = threadIdx.x + blockIdx.x * blockDim.x;
    num_threads = gridDim.x * blockDim.x;
    
    gap = (extent - size) / 8;
    nb_elements = size / 8;
    _dst_disp_tmp = (double*)destination;
    _source_tmp = (double*)source;
    _destination_tmp = _dst_disp_tmp + tid;
    _source_tmp += tid;

    for (_i = tid; _i < copy_loops*nb_elements; _i+=num_threads) {
        _destination_tmp = _dst_disp_tmp + tid + _i/num_threads*num_threads + _i/nb_elements * gap;
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
        // if (_i % nb_elements == 0 ) {
        //     DBGPRINT("tid %d, pack 3. memcpy( %p, %p, %lu ) => _i %d, actual _i %d\n",
        //                                     tid, _destination_tmp, _source_tmp, (unsigned long)size,  _i/nb_elements, _i );
        // }
        // if (_i / nb_elements ==1 && tid == 0 ) {
        //     DBGPRINT("tid %d, pack 3. memcpy( %p, %p, %lu ) => space %lu, _i %d, actual _i %d\n",
        //                                     tid, _destination_tmp, _source_tmp, (unsigned long)_end_loop->size, (unsigned long)(*(SPACE) - _i/nb_elements * _end_loop->size), _i/nb_elements, _i );
        // }
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
#if !defined (OPAL_DATATYPE_CUDA_DRY_RUN)
        *_destination_tmp = *_source_tmp;
#endif /* ! OPAL_DATATYPE_CUDA_DRY_RUN */
        _source_tmp += num_threads;
    }
}
