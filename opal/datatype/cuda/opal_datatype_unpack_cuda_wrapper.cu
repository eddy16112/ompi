#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"

#include <stdio.h>
#include <assert.h>

int32_t opal_generic_simple_unpack_function_cuda( opal_convertor_t* pConvertor,
                                                  struct iovec* iov, 
                                                  uint32_t* out_size,
                                                  size_t* max_data )
{
    uint32_t i;
    dt_elem_desc_t* description;
    const opal_datatype_t *pData = pConvertor->pDesc;
    uint32_t tasks_per_block, num_blocks, thread_per_block;
    dt_stack_t* pStack;
    
    return -99;
    description = pConvertor->use_desc->desc;
    
    cuda_desc_h->stack_pos = pConvertor->stack_pos;
#if defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    cuda_desc_h->pBaseBuf = pConvertor->pBaseBuf;
#else
    cuda_desc_h->pBaseBuf = pBaseBuf_GPU;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
    cuda_desc_h->lb = pData->lb;
    cuda_desc_h->ub = pData->ub;
    cuda_desc_h->out_size = *out_size;
    cuda_desc_h->max_data = *max_data;
    cuda_desc_h->bConverted = pConvertor->bConverted;
    cuda_desc_h->local_size = pConvertor->local_size;
    cuda_desc_h->stack_size = pConvertor->stack_size;
    
    for (i = 0; i < pConvertor->stack_size; i++) {
        cuda_desc_h->pStack[i] = pConvertor->pStack[i];
    }
    if (cuda_desc_h->description_max_count != 0) {
        if (cuda_desc_h->description_max_count >= (pConvertor->use_desc->used+1)) {
            cuda_desc_h->description_count = pConvertor->use_desc->used+1;
        } else {
            hipFree(cuda_desc_h->description);
            cuda_desc_h->description = NULL;
            hipMalloc((void **)&(cuda_desc_h->description), sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1));
            cuda_desc_h->description_max_count = pConvertor->use_desc->used+1;
            cuda_desc_h->description_count = pConvertor->use_desc->used+1;
        }
        
    } else {
        hipMalloc((void **)&(cuda_desc_h->description), sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1));
        cuda_desc_h->description_max_count = pConvertor->use_desc->used+1;
        cuda_desc_h->description_count = pConvertor->use_desc->used+1;
    }
    hipMemcpy(cuda_desc_h->description, description, sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1), hipMemcpyHostToDevice);
    
    DBGPRINT("stack_size %d\n", pConvertor->stack_size);

    DBGPRINT("flags %d, types %d, count %d\n", description->elem.common.flags, description->elem.common.type, description->elem.count);
    
    for (i = 0; i < *out_size; i++) {
#if defined (OPAL_DATATYPE_CUDA_DRY_RUN)
        cuda_desc_h->iov[i].iov_base = iov[i].iov_base;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
        cuda_desc_h->iov[i].iov_len = iov[i].iov_len;
    }
    
    hipMemcpy(cuda_desc_d, cuda_desc_h, sizeof(ddt_cuda_desc_t), hipMemcpyHostToDevice);
    
    pStack = pConvertor->pStack + pConvertor->stack_pos;
    thread_per_block = CUDA_WARP_SIZE * 3;
    tasks_per_block = thread_per_block * TASK_PER_THREAD;
    num_blocks = ((uint32_t)pStack->count + tasks_per_block - 1) / tasks_per_block;
    printf("launch unpack kernel, count %d, num_blocks %d, total threads %d\n", (uint32_t)pStack->count, num_blocks, num_blocks*thread_per_block);
    opal_generic_simple_unpack_cuda_kernel<<<192, thread_per_block>>>(cuda_desc_d);
#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    size_t position = pConvertor->pDesc->size;
    opal_convertor_set_position_nocheck(pConvertor, &position);
#endif
    hipDeviceSynchronize();
    
#if defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    return -99;
#else
    // /* copy stack and description data back to CPU */
    // hipMemcpy(cuda_desc_h, cuda_desc_d, sizeof(ddt_cuda_desc_t), hipMemcpyDeviceToHost);
    //
    // for (i = 0; i < pConvertor->stack_size; i++) {
    //     pConvertor->pStack[i] = cuda_desc_h->pStack[i];
    // }
    //
    // pConvertor->stack_pos = cuda_desc_h->stack_pos;
    // *out_size = cuda_desc_h->out_size;
    // *max_data = cuda_desc_h->max_data;
    // pConvertor->bConverted = cuda_desc_h->bConverted;
    // pConvertor->local_size = cuda_desc_h->local_size;
    //
    // for (i = 0; i < *out_size; i++) {
    //     iov[i].iov_len = cuda_desc_h->iov[i].iov_len;
    // }
    //
    if( pConvertor->bConverted == pConvertor->local_size ) {
        // pConvertor->flags |= CONVERTOR_COMPLETED;
        return 1;
    }

    return 0;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
}

int32_t opal_generic_simple_unpack_function_cuda_iov( opal_convertor_t* pConvertor,
                                                  struct iovec* iov, 
                                                  uint32_t* out_size,
                                                  size_t* max_data )
{
    uint32_t i, j;
    uint32_t count_desc, current_block, task_iteration, nb_blocks_per_description, dst_offset, residue_desc;
    uint32_t nb_blocks, thread_per_block;
    size_t length, buffer_size, length_per_iovec;
    unsigned char *source;
    size_t total_unpacked, total_converted;
    int32_t complete_flag = 0;
    uint8_t buffer_isfull = 0;
    uint32_t convertor_flags;
    dt_elem_desc_t* description;
    dt_elem_desc_t* pElem;
    dt_stack_t* pStack;
    uint8_t alignment, orig_alignment;
    
    ddt_cuda_iov_dist_t* cuda_iov_dist_h_current;
    ddt_cuda_iov_dist_t* cuda_iov_dist_d_current;
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time;
#endif
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start_total);
#endif
    
    description = pConvertor->use_desc->desc;
    pStack = pConvertor->pStack + pConvertor->stack_pos;
    pElem = &(description[pStack->index]);
    printf("size elem %lu, size %d\n", pElem->elem.common.type, opal_datatype_basicDatatypesSize[pElem->elem.common.type]);
    
    DT_CUDA_DEBUG ( opal_cuda_output(0, "GPU datatype UNpacking using iovec\n"); );
    
#if defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    source = (unsigned char*)iov[0].iov_base;
#else
//    pConvertor->pBaseBuf = pBaseBuf_GPU;
 //   printf("Unpack GPU base %p, iov buffer %p\n", pConvertor->pBaseBuf, iov[0].iov_base);
    source = ddt_cuda_unpack_buffer;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
    
    // double *vtmp = (double *)iov[0].iov_base;
    printf("recevied unpacked iov buffer, len %d\n", iov[0].iov_len);
    // for (uint32_t i = 0; i < iov[0].iov_len/sizeof(double); i++) {
    //     printf(" %1.f ", *vtmp);
    //     vtmp ++;
    // }
    // printf("\n");
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif    
    hipMemcpy(source, iov[0].iov_base, iov[0].iov_len, hipMemcpyHostToDevice);
#if defined(OPAL_DATATYPE_CUDA_TIMING) 
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    printf( "[Timing]: HtoD memcpy in %ld microsec\n", total_time );
#endif


#if defined (OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    buffer_size = iov[0].iov_len;
    cuda_iov_count = 1000;
    total_unpacked = 0;
    total_converted = pConvertor->bConverted;
    cuda_streams->current_stream_id = 0;
    convertor_flags = pConvertor->flags;
    complete_flag = opal_convertor_raw( pConvertor, cuda_iov, &cuda_iov_count, &length );
    DT_CUDA_DEBUG ( opal_cuda_output(1, "complete flag %d, iov count %d, length %d, submit to CUDA stream %d\n", complete_flag, cuda_iov_count, length, cuda_streams->current_stream_id); );
    
#if defined (OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    printf( "[Timing]: ddt to iov in %ld microsec\n", total_time );
#endif
    
    dst_offset = 0;
    thread_per_block = CUDA_WARP_SIZE * 4;
    nb_blocks = 256;
    
    while (cuda_iov_count > 0) {
        
        current_block = 0;
        task_iteration = 0;
        cuda_iov_dist_h_current = cuda_iov_dist_h[cuda_streams->current_stream_id];
        cuda_iov_dist_d_current = cuda_iov_dist_d[cuda_streams->current_stream_id]; 
        
#if defined (OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        for (i = 0; i < nb_blocks; i++) {
            cuda_iov_dist_h_current[i].nb_tasks = 0;
        }
        
        for (i = 0; i < cuda_iov_count; i++) {
            if (buffer_size >= cuda_iov[i].iov_len) {
                length_per_iovec = cuda_iov[i].iov_len;
            } else {
                orig_alignment = opal_datatype_basicDatatypesSize[pElem->elem.common.type];
                length_per_iovec = buffer_size / orig_alignment * orig_alignment;
                buffer_isfull = 1;
            }
            buffer_size -= length_per_iovec;
            total_unpacked += length_per_iovec;
            
            /* check alignment */
            if ((uintptr_t)(cuda_iov[i].iov_base) % ALIGNMENT_DOUBLE == 0 && (uintptr_t)source % ALIGNMENT_DOUBLE == 0) {
                alignment = ALIGNMENT_DOUBLE;
            } else if ((uintptr_t)(cuda_iov[i].iov_base) % ALIGNMENT_FLOAT == 0 && (uintptr_t)source % ALIGNMENT_FLOAT == 0) {
                alignment = ALIGNMENT_FLOAT;
            } else {
                alignment = ALIGNMENT_CHAR;
            }
            
           // alignment = ALIGNMENT_CHAR;

            count_desc = length_per_iovec / alignment;
            residue_desc = length_per_iovec % alignment;
            nb_blocks_per_description = (count_desc + thread_per_block - 1) / thread_per_block;
            DT_CUDA_DEBUG ( opal_cuda_output(2, "description %d, size %d, residue %d, alignment %d\n", i, count_desc, residue_desc, alignment); );
            for (j = 0; j < nb_blocks_per_description; j++) {
                cuda_iov_dist_h_current[current_block].dst[task_iteration] = (unsigned char *)(cuda_iov[i].iov_base) + j * thread_per_block * alignment;
                cuda_iov_dist_h_current[current_block].src[task_iteration] = source;
                cuda_iov_dist_h_current[current_block].element_alignment[task_iteration] = alignment;
                cuda_iov_dist_h_current[current_block].nb_tasks = task_iteration + 1;
                if ( (j+1) * thread_per_block <= count_desc) {
                    cuda_iov_dist_h_current[current_block].nb_elements[task_iteration] = thread_per_block;// * sizeof(double);
                } else {
                    cuda_iov_dist_h_current[current_block].nb_elements[task_iteration] = (thread_per_block - ((j+1)*thread_per_block - count_desc));// * sizeof(double);
                }
                source += cuda_iov_dist_h_current[current_block].nb_elements[task_iteration] * alignment;
                DT_CUDA_DEBUG( opal_cuda_output(3, "\tblock %d, src %p, dst %p, nb_elements %d, alignment %d\n", current_block, cuda_iov_dist_h_current[current_block].src[task_iteration], cuda_iov_dist_h_current[current_block].dst[task_iteration], cuda_iov_dist_h_current[current_block].nb_elements[task_iteration], cuda_iov_dist_h_current[current_block].element_alignment[task_iteration]); );
                current_block += 1;
                if (current_block >= nb_blocks) {
                    current_block = 0;
                    task_iteration ++;
                    assert(task_iteration < CUDA_IOV_MAX_TASK_PER_BLOCK);
                }
            }
            
            /* handle residue */
            if (residue_desc != 0) {
                orig_alignment = opal_datatype_basicDatatypesSize[pElem->elem.common.type];
                cuda_iov_dist_h_current[current_block].dst[task_iteration] = (unsigned char *)(cuda_iov[i].iov_base) + length_per_iovec / alignment * alignment;
                cuda_iov_dist_h_current[current_block].src[task_iteration] = source;
                cuda_iov_dist_h_current[current_block].element_alignment[task_iteration] = orig_alignment;
                cuda_iov_dist_h_current[current_block].nb_tasks = task_iteration + 1;
                cuda_iov_dist_h_current[current_block].nb_elements[task_iteration] = (length_per_iovec - length_per_iovec / alignment * alignment) / orig_alignment;
                source += cuda_iov_dist_h_current[current_block].nb_elements[task_iteration] * orig_alignment;
                DT_CUDA_DEBUG( opal_cuda_output(3, "\tblock %d, src %p, dst %p, nb_elements %d, alignment %d\n", current_block, cuda_iov_dist_h_current[current_block].src[task_iteration], cuda_iov_dist_h_current[current_block].dst[task_iteration], cuda_iov_dist_h_current[current_block].nb_elements[task_iteration], cuda_iov_dist_h_current[current_block].element_alignment[task_iteration]); );
                current_block += 1;
                if (current_block >= nb_blocks) {
                    current_block = 0;
                    task_iteration ++;
                    assert(task_iteration < CUDA_IOV_MAX_TASK_PER_BLOCK);
                }
            }
            
            if (buffer_isfull) {
                break;
            }
        }

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        printf( "[Timing]: iov is prepared in %ld microsec, hipMemcpy will be submit to CUDA stream %d\n", total_time,  cuda_streams->current_stream_id);
#endif
                
        hipMemcpyAsync(cuda_iov_dist_d_current, cuda_iov_dist_h_current, sizeof(ddt_cuda_iov_dist_t)*(nb_blocks), hipMemcpyHostToDevice, cuda_streams->opal_cuda_stream[cuda_streams->current_stream_id]);
        opal_generic_simple_unpack_cuda_iov_kernel<<<nb_blocks, thread_per_block, 0, cuda_streams->opal_cuda_stream[cuda_streams->current_stream_id]>>>(cuda_iov_dist_d_current);
        cuda_streams->current_stream_id ++;
        cuda_streams->current_stream_id = cuda_streams->current_stream_id % NB_STREAMS;    
        
        /* buffer is full */
        if (buffer_isfull) {
            pConvertor->flags = convertor_flags;
            total_converted += total_unpacked;
            opal_convertor_set_position_nocheck(pConvertor, &total_converted);
            break;
        }
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif   
        convertor_flags = pConvertor->flags;     
        complete_flag = opal_convertor_raw( pConvertor, cuda_iov, &cuda_iov_count, &length );
        DT_CUDA_DEBUG ( opal_cuda_output(1, "complete flag %d, iov count %d, length %d, submit to CUDA stream %d\n", complete_flag, cuda_iov_count, length, cuda_streams->current_stream_id); );
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        printf( "[Timing]: ddt to iov in %ld microsec\n", total_time );
#endif

    }
    hipDeviceSynchronize();
    
    iov[0].iov_len = total_unpacked;
    *max_data = total_unpacked;
    *out_size = 1;
    
    DT_CUDA_DEBUG ( opal_cuda_output(0, "total unpacked %d\n", total_unpacked); );
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end_total );
    total_time = ELAPSED_TIME( start_total, end_total );
    printf( "[Timing]: total unpacking in %ld microsec\n", total_time );
#endif
    
    if( pConvertor->bConverted == pConvertor->local_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        return 1;
    }        
    return 0;   
}

void unpack_contiguous_loop_cuda( dt_elem_desc_t* ELEM,
                                  uint32_t* COUNT,
                                  unsigned char** SOURCE,
                                  unsigned char** DESTINATION,
                                  size_t* SPACE )
{
    ddt_loop_desc_t *_loop = (ddt_loop_desc_t*)(ELEM);
    ddt_endloop_desc_t* _end_loop = (ddt_endloop_desc_t*)((ELEM) + _loop->items);
    unsigned char* _destination = (*DESTINATION) + _end_loop->first_elem_disp;
    uint32_t _copy_loops = *(COUNT);
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _source = *(SOURCE);

    printf("I am in unpack_contiguous_loop_cuda\n");

    if( (_copy_loops * _end_loop->size) > *(SPACE) )
        _copy_loops = (uint32_t)(*(SPACE) / _end_loop->size);

    _destination = pBaseBuf_GPU;
    _source = (unsigned char*)cuda_desc_h->iov[0].iov_base;
    
    tasks_per_block = THREAD_PER_BLOCK * TASK_PER_THREAD;
    num_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;
    unpack_contiguous_loop_cuda_kernel_global<<<192, 4*THREAD_PER_BLOCK>>>(_copy_loops, _end_loop->size, _loop->extent, _source, _destination);
    
    *(DESTINATION) = _destination - _end_loop->first_elem_disp;
    *(SPACE) -= _copy_loops * _end_loop->size;
    *(COUNT) -= _copy_loops;
    
    hipDeviceSynchronize();
}
