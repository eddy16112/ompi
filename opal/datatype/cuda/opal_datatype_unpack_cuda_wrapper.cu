#include "hip/hip_runtime.h"
#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"

#include <stdio.h>
#include <assert.h>


int32_t opal_ddt_generic_simple_unpack_function_cuda_vector( opal_convertor_t* pConvertor,
                                                         struct iovec* iov, uint32_t* out_size,
                                                         size_t* max_data )
{
    dt_stack_t* pStack;                /* pointer to the position on the stack */
    uint32_t pos_desc;                 /* actual position in the description of the derived datatype */
    uint32_t count_desc;               /* the number of items already done in the actual pos_desc */
    size_t total_unpacked = 0;         /* total size unpacked this time */
    dt_elem_desc_t* description;
    dt_elem_desc_t* pElem;
    const opal_datatype_t *pData = pConvertor->pDesc;
    unsigned char *conv_ptr, *iov_ptr;
    size_t iov_len_local;
    uint32_t iov_count;
    uint8_t free_required;
    uint32_t count_desc_tmp;
    
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    int contiguous_loop_flag = 0;
    int i;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif

    DT_CUDA_DEBUG( opal_cuda_output( 2, "opal_convertor_generic_simple_unpack( %p, {%p, %lu}, %u , %u)\n",
                                     (void*)pConvertor, iov[0].iov_base, (unsigned long)iov[0].iov_len, *out_size, *max_data ); )

    description = pConvertor->use_desc->desc;

    /* For the first step we have to add both displacement to the source. After in the
     * main while loop we will set back the source_base to the correct value. This is
     * due to the fact that the convertor can stop in the middle of a data with a count
     */
    pStack     = pConvertor->pStack + pConvertor->stack_pos;
    pos_desc   = pStack->index;
    conv_ptr   = pConvertor->pBaseBuf + pStack->disp;
    count_desc = (uint32_t)pStack->count;
    pStack--;
    pConvertor->stack_pos--;
    pElem = &(description[pos_desc]);

    DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack start pos_desc %d count_desc %d disp %ld\n"
                           "stack_pos %d pos_desc %d count_desc %d disp %ld\n",
                           pos_desc, count_desc, (long)(conv_ptr - pConvertor->pBaseBuf),
                           pConvertor->stack_pos, pStack->index, (int)pStack->count, (long)(pStack->disp) ); );

    for( iov_count = 0; iov_count < (*out_size); iov_count++ ) {
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        if (opal_ddt_cuda_is_gpu_buffer(iov[iov_count].iov_base)) {
            iov_ptr = (unsigned char*)iov[iov_count].iov_base;
            free_required = 0;
        } else {
            if (OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_D2H || OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
                iov_ptr = (unsigned char*)iov[iov_count].iov_base;
                pConvertor->gpu_buffer_ptr = NULL;
                free_required = 0;
            } else {
                if (pConvertor->gpu_buffer_ptr == NULL) {
                    pConvertor->gpu_buffer_ptr = (unsigned char*)opal_ddt_cuda_malloc_gpu_buffer(iov[iov_count].iov_len, 0);
                }
                iov_ptr = pConvertor->gpu_buffer_ptr;
                hipMemcpy(iov_ptr, iov[iov_count].iov_base, iov[iov_count].iov_len, hipMemcpyHostToDevice);
                free_required = 1;
            }
        } 
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG( opal_cuda_output( 2, "[Timing]: HtoD memcpy in %ld microsec, free required %d\n", total_time, free_required ); );
#endif
        iov_len_local = iov[iov_count].iov_len;
        hipDeviceSynchronize();
        if( 0 != pConvertor->partial_length ) {
            /* not support yet */
        }
        while( 1 ) {
            while( pElem->elem.common.flags & OPAL_DATATYPE_FLAG_DATA ) {
                /* now here we have a basic datatype */
                /* should not go to here */
                unpack_predefined_data_cuda( pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local );
                if( 0 == count_desc ) {  /* completed */
                    conv_ptr = pConvertor->pBaseBuf + pStack->disp;
                    pos_desc++;  /* advance to the next data */
                    UPDATE_INTERNAL_COUNTERS( description, pos_desc, pElem, count_desc );
                    continue;
                }
                if (contiguous_loop_flag) {
                    pStack--;
                    pConvertor->stack_pos--;
                    pos_desc --;
                    pElem = &(description[pos_desc]);
                    count_desc = count_desc_tmp;
                }
                assert( pElem->elem.common.type < OPAL_DATATYPE_MAX_PREDEFINED );
                if( 0 != iov_len_local ) {
                    assert(0);
                }
                goto complete_loop;
            }
            if( OPAL_DATATYPE_END_LOOP == pElem->elem.common.type ) { /* end of the current loop */
                DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack end_loop count %d stack_pos %d pos_desc %d disp %ld space %lu\n",
                                                 (int)pStack->count, pConvertor->stack_pos, pos_desc,
                                                 (long)pStack->disp, (unsigned long)iov_len_local ); );
                if( --(pStack->count) == 0 ) { /* end of loop */
                    if( 0 == pConvertor->stack_pos ) {
                        /* Do the same thing as when the loop is completed */
                        iov[iov_count].iov_len -= iov_len_local;  /* update the amount of valid data */
                        total_unpacked += iov[iov_count].iov_len;
                        iov_count++;  /* go to the next */
                        goto complete_conversion;
                    }
                    pConvertor->stack_pos--;
                    pStack--;
                    pos_desc++;
                } else {
                    pos_desc = pStack->index + 1;
                    if( pStack->index == -1 ) {
                        pStack->disp += (pData->ub - pData->lb);
                    } else {
                        assert( OPAL_DATATYPE_LOOP == description[pStack->index].loop.common.type );
                        pStack->disp += description[pStack->index].loop.extent;
                    }
                }
                conv_ptr = pConvertor->pBaseBuf + pStack->disp;
                UPDATE_INTERNAL_COUNTERS( description, pos_desc, pElem, count_desc );
                DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack new_loop count %d stack_pos %d pos_desc %d disp %ld space %lu\n",
                                                 (int)pStack->count, pConvertor->stack_pos, pos_desc,
                                                 (long)pStack->disp, (unsigned long)iov_len_local ); );
            }
            if( OPAL_DATATYPE_LOOP == pElem->elem.common.type ) {
                OPAL_PTRDIFF_TYPE local_disp = (OPAL_PTRDIFF_TYPE)conv_ptr;
                if( pElem->loop.common.flags & OPAL_DATATYPE_FLAG_CONTIGUOUS ) {
                    if (OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_D2H) {
                        unpack_contiguous_loop_cuda_memcpy2d_d2h(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    } else if (OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
                        unpack_contiguous_loop_cuda_zerocopy(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    } else {
                        unpack_contiguous_loop_cuda(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    }
                    if( 0 == count_desc ) {  /* completed */
                        pos_desc += pElem->loop.items + 1;
                        goto update_loop_description;
                    } else {
                        contiguous_loop_flag = 1;
                    }
                    /* Save the stack with the correct last_count value. */
                }
                local_disp = (OPAL_PTRDIFF_TYPE)conv_ptr - local_disp;
                PUSH_STACK( pStack, pConvertor->stack_pos, pos_desc, OPAL_DATATYPE_LOOP, count_desc,
                            pStack->disp + local_disp);
                pos_desc++;
            update_loop_description:  /* update the current state */
                if (contiguous_loop_flag) {
                    count_desc_tmp = count_desc;
                } else {
                    conv_ptr = pConvertor->pBaseBuf + pStack->disp;
                }
                UPDATE_INTERNAL_COUNTERS( description, pos_desc, pElem, count_desc );
                continue;
            }
        }
    complete_loop:
        iov[iov_count].iov_len -= iov_len_local;  /* update the amount of valid data */
        total_unpacked += iov[iov_count].iov_len;
    }
 complete_conversion:
    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
    *max_data = total_unpacked;
    pConvertor->bConverted += total_unpacked;  /* update the already converted bytes */
    *out_size = iov_count;
    if( pConvertor->bConverted == pConvertor->remote_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Unpack total unpacked %lu\n", pConvertor->bConverted); );
        if (pConvertor->gpu_buffer_ptr != NULL && free_required == 1) {
            opal_ddt_cuda_free_gpu_buffer(pConvertor->gpu_buffer_ptr, 0);
            pConvertor->gpu_buffer_ptr = NULL;
        }
        return 1;
    }
    /* Save the global position for the next round */
    PUSH_STACK( pStack, pConvertor->stack_pos, pos_desc, pElem->elem.common.type, count_desc,
                conv_ptr - pConvertor->pBaseBuf );
    DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack save stack stack_pos %d pos_desc %d count_desc %d disp %ld\n",
                                     pConvertor->stack_pos, pStack->index, (int)pStack->count, (long)pStack->disp ); );
    return 0;
}

int32_t opal_ddt_generic_simple_unpack_function_cuda_vector2( opal_convertor_t* pConvertor,
                                                         struct iovec* iov, uint32_t* out_size,
                                                         size_t* max_data )
{
    dt_stack_t* pStack;                /* pointer to the position on the stack */
    uint32_t pos_desc;                 /* actual position in the description of the derived datatype */
    uint32_t count_desc;               /* the number of items already done in the actual pos_desc */
    size_t total_unpacked = 0;         /* total size unpacked this time */
    dt_elem_desc_t* description;
    dt_elem_desc_t* pElem;
    const opal_datatype_t *pData = pConvertor->pDesc;
    unsigned char *conv_ptr, *iov_ptr;
    size_t iov_len_local;
    uint32_t iov_count;
    uint8_t free_required;
    uint32_t count_desc_tmp;
    
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif

    DT_CUDA_DEBUG( opal_cuda_output( 2, "opal_convertor_generic_simple_unpack_vector( %p, {%p, %lu}, %u , %u)\n",
                                     (void*)pConvertor, iov[0].iov_base, (unsigned long)iov[0].iov_len, *out_size, *max_data ); )

    description = pConvertor->use_desc->desc;

    /* For the first step we have to add both displacement to the source. After in the
     * main while loop we will set back the source_base to the correct value. This is
     * due to the fact that the convertor can stop in the middle of a data with a count
     */
    pStack     = pConvertor->pStack + pConvertor->stack_pos;
    pos_desc   = pStack->index;
    conv_ptr   = pConvertor->pBaseBuf + pStack->disp;
    count_desc = (uint32_t)pStack->count;
    pStack--;
    pConvertor->stack_pos--;
    pElem = &(description[pos_desc]);

    DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack start pos_desc %d count_desc %d disp %ld\n"
                           "stack_pos %d pos_desc %d count_desc %d disp %ld\n",
                           pos_desc, count_desc, (long)(conv_ptr - pConvertor->pBaseBuf),
                           pConvertor->stack_pos, pStack->index, (int)pStack->count, (long)(pStack->disp) ); );

    for( iov_count = 0; iov_count < (*out_size); iov_count++ ) {
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        if (opal_ddt_cuda_is_gpu_buffer(iov[iov_count].iov_base)) {
            iov_ptr = (unsigned char*)iov[iov_count].iov_base;
            free_required = 0;
        } else {
            if (OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_D2H || OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
                iov_ptr = (unsigned char*)iov[iov_count].iov_base;
                pConvertor->gpu_buffer_ptr = NULL;
                free_required = 0;
            } else {
                if (pConvertor->gpu_buffer_ptr == NULL) {
                    pConvertor->gpu_buffer_ptr = (unsigned char*)opal_ddt_cuda_malloc_gpu_buffer(iov[iov_count].iov_len, 0);
                }
                iov_ptr = pConvertor->gpu_buffer_ptr;
                hipMemcpy(iov_ptr, iov[iov_count].iov_base, iov[iov_count].iov_len, hipMemcpyHostToDevice);
                free_required = 1;
            }
        }
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG( opal_cuda_output( 2, "[Timing]: HtoD memcpy in %ld microsec, free required %d\n", total_time, free_required ); );
#endif
        iov_len_local = iov[iov_count].iov_len;
        if( 0 != pConvertor->partial_length ) {
            /* not support yet */
        }
        while( 1 ) {
            while( pElem->elem.common.flags & OPAL_DATATYPE_FLAG_DATA ) {
                /* now here we have a basic datatype */
                /* should not go to here */
                pStack--;
                pConvertor->stack_pos--;
                pos_desc --;
                pElem = &(description[pos_desc]);
                count_desc = count_desc_tmp;
                goto complete_loop;
            }
            if( OPAL_DATATYPE_END_LOOP == pElem->elem.common.type ) { /* end of the current loop */
                DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack end_loop count %d stack_pos %d pos_desc %d disp %ld space %lu\n",
                                                 (int)pStack->count, pConvertor->stack_pos, pos_desc,
                                                 (long)pStack->disp, (unsigned long)iov_len_local ); );
                if( --(pStack->count) == 0 ) { /* end of loop */
                    if( 0 == pConvertor->stack_pos ) {
                        /* Do the same thing as when the loop is completed */
                        iov[iov_count].iov_len -= iov_len_local;  /* update the amount of valid data */
                        total_unpacked += iov[iov_count].iov_len;
                        iov_count++;  /* go to the next */
                        goto complete_conversion;
                    }
                    pConvertor->stack_pos--;
                    pStack--;
                    pos_desc++;
                } else {
                    pos_desc = pStack->index + 1;
                    if( pStack->index == -1 ) {
                        pStack->disp += (pData->ub - pData->lb);
                    } else {
                        assert( OPAL_DATATYPE_LOOP == description[pStack->index].loop.common.type );
                        pStack->disp += description[pStack->index].loop.extent;
                    }
                }
                conv_ptr = pConvertor->pBaseBuf + pStack->disp;
                UPDATE_INTERNAL_COUNTERS( description, pos_desc, pElem, count_desc );
                DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack new_loop count %d stack_pos %d pos_desc %d disp %ld space %lu\n",
                                                 (int)pStack->count, pConvertor->stack_pos, pos_desc,
                                                 (long)pStack->disp, (unsigned long)iov_len_local ); );
            }
            if( OPAL_DATATYPE_LOOP == pElem->elem.common.type ) {
                OPAL_PTRDIFF_TYPE local_disp = (OPAL_PTRDIFF_TYPE)conv_ptr;
                if( pElem->loop.common.flags & OPAL_DATATYPE_FLAG_CONTIGUOUS ) {
                    if (OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_D2H) {
                        unpack_contiguous_loop_cuda_memcpy2d_d2h(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    } else if (OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
                        unpack_contiguous_loop_cuda_zerocopy(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    } else {
                        unpack_contiguous_loop_cuda(pElem, &count_desc, &iov_ptr, &conv_ptr, &iov_len_local);
                    }
                    if( 0 == count_desc ) {  /* completed */
                        pos_desc += pElem->loop.items + 1;
                        goto update_loop_description;
                    }
                    /* Save the stack with the correct last_count value. */
                }
                local_disp = (OPAL_PTRDIFF_TYPE)conv_ptr - local_disp;
                PUSH_STACK( pStack, pConvertor->stack_pos, pos_desc, OPAL_DATATYPE_LOOP, count_desc,
                            pStack->disp + local_disp);
                pos_desc++;
            update_loop_description:  /* update the current state */
            //    conv_ptr = pConvertor->pBaseBuf + pStack->disp;
                count_desc_tmp = count_desc;
                UPDATE_INTERNAL_COUNTERS( description, pos_desc, pElem, count_desc );
                continue;
            }
        }
    complete_loop:
        iov[iov_count].iov_len -= iov_len_local;  /* update the amount of valid data */
        total_unpacked += iov[iov_count].iov_len;
    }
 complete_conversion:
    *max_data = total_unpacked;
    pConvertor->bConverted += total_unpacked;  /* update the already converted bytes */
    *out_size = iov_count;
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Unpack total unpacked %lu\n", total_unpacked); );
    if( pConvertor->bConverted == pConvertor->remote_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        if (pConvertor->gpu_buffer_ptr != NULL && free_required == 1) {
            opal_ddt_cuda_free_gpu_buffer(pConvertor->gpu_buffer_ptr, 0);
            pConvertor->gpu_buffer_ptr = NULL;
        }
        return 1;
    }
    /* Save the global position for the next round */
    PUSH_STACK( pStack, pConvertor->stack_pos, pos_desc, pElem->elem.common.type, count_desc,
                conv_ptr - pConvertor->pBaseBuf );
    DT_CUDA_DEBUG( opal_cuda_output( 4, "unpack save stack stack_pos %d pos_desc %d count_desc %d disp %ld\n",
                                     pConvertor->stack_pos, pStack->index, (int)pStack->count, (long)pStack->disp ); );
    return 0;
}


int32_t opal_ddt_generic_simple_unpack_function_cuda_iov( opal_convertor_t* pConvertor,
                                                          struct iovec* iov,
                                                          uint32_t* out_size,
                                                          size_t* max_data )
{
    size_t buffer_size;
    unsigned char *source;
    size_t total_unpacked;
    uint8_t free_required = 0;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time, move_time;
#endif

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start_total);
#endif

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    if (opal_ddt_cuda_is_gpu_buffer(iov[0].iov_base)) {
        source = (unsigned char*)iov[0].iov_base;
        free_required = 0;
    } else {
        if (OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
            hipHostGetDevicePointer((void **)&source, (void *)iov[0].iov_base, 0);
            pConvertor->gpu_buffer_ptr = NULL;
            free_required = 0;
        } else {
            if (pConvertor->gpu_buffer_ptr == NULL) {
                pConvertor->gpu_buffer_ptr = (unsigned char*)opal_ddt_cuda_malloc_gpu_buffer(iov[0].iov_len, 0);
            }
            source = pConvertor->gpu_buffer_ptr;
            hipMemcpy(source, iov[0].iov_base, iov[0].iov_len, hipMemcpyHostToDevice);
            free_required = 1;
        }
    }

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end );
    move_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: HtoD memcpy in %ld microsec, free required %d\n", move_time, free_required ); );
#endif


    buffer_size = iov[0].iov_len;
    total_unpacked = 0;
    
    /* start unpack */
    if (cuda_iov_cache_enabled) {
        opal_ddt_generic_simple_unpack_function_cuda_iov_cached(pConvertor, source, buffer_size, &total_unpacked);
    } else {
        opal_ddt_generic_simple_unpack_function_cuda_iov_non_cached(pConvertor, source, buffer_size, &total_unpacked);
    }
    
    pConvertor->bConverted += total_unpacked;
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack total unpacked %d\n", total_unpacked); );

    iov[0].iov_len = total_unpacked;
    *max_data = total_unpacked;
    *out_size = 1;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end_total );
    total_time = ELAPSED_TIME( start_total, end_total );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: total unpacking in %ld microsec, kernel %ld microsec\n", total_time, total_time-move_time ); );
#endif

    if( pConvertor->bConverted == pConvertor->local_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        if (pConvertor->gpu_buffer_ptr != NULL && free_required) {
            opal_ddt_cuda_free_gpu_buffer(pConvertor->gpu_buffer_ptr, 0);
            pConvertor->gpu_buffer_ptr = NULL;
        }
        return 1;
    }
    return 0;
}

#if 0
int32_t opal_ddt_generic_simple_unpack_function_cuda_iov_non_cached( opal_convertor_t* pConvertor,
                                                                     struct iovec* iov,
                                                                     uint32_t* out_size,
                                                                     size_t* max_data )
{
    uint32_t i, j;
    uint32_t count_desc, nb_blocks_per_description, dst_offset, residue_desc;
    uint32_t nb_blocks, thread_per_block, nb_blocks_used;
    size_t length, buffer_size, length_per_iovec;
    unsigned char *source, *source_base;
    size_t total_unpacked, total_converted;
    int32_t complete_flag = 0;
    uint8_t buffer_isfull = 0;
    uint8_t free_required = 0;
    uint32_t convertor_flags;
//    dt_elem_desc_t* description;
//    dt_elem_desc_t* pElem;
//    dt_stack_t* pStack;
    uint8_t alignment, orig_alignment;
//    int32_t orig_stack_index;
    hipError_t cuda_err;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    ddt_cuda_iov_dist_non_cached_t* cuda_iov_dist_h_current;
    ddt_cuda_iov_dist_non_cached_t* cuda_iov_dist_d_current;
    ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block;
    int iov_pipeline_block_id = 0;
    hipStream_t *cuda_stream_iov = NULL;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time, move_time;
#endif

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start_total);
#endif

/*    description = pConvertor->use_desc->desc;
    pStack = pConvertor->pStack + pConvertor->stack_pos;
    pElem = &(description[pStack->index]);
    printf("size elem %d, size %lu\n", pElem->elem.common.type, opal_datatype_basicDatatypes[pElem->elem.common.type]->size);
*/

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    if (opal_ddt_cuda_is_gpu_buffer(iov[0].iov_base)) {
        source = (unsigned char*)iov[0].iov_base;
        free_required = 0;
    } else {
        if (OPAL_DATATYPE_VECTOR_USE_ZEROCPY) {
            hipHostGetDevicePointer((void **)&source, (void *)iov[0].iov_base, 0);
            pConvertor->gpu_buffer_ptr = NULL;
            free_required = 0;
        } else {
            if (pConvertor->gpu_buffer_ptr == NULL) {
                pConvertor->gpu_buffer_ptr = (unsigned char*)opal_ddt_cuda_malloc_gpu_buffer(iov[0].iov_len, 0);
            }
            source = pConvertor->gpu_buffer_ptr;
            hipMemcpy(source, iov[0].iov_base, iov[0].iov_len, hipMemcpyHostToDevice);
            free_required = 1;
        }
    }

    source_base = source;
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack using IOV non cached, GPU base %p, unpack from buffer %p, total size %ld\n",
                                     pConvertor->pBaseBuf, source, iov[0].iov_len); );
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end );
    move_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: HtoD memcpy in %ld microsec, free required %d\n", move_time, free_required ); );
#endif
    
//    cuda_err = hipEventRecord(current_cuda_device->memcpy_event, current_cuda_device->cuda_streams->opal_cuda_stream[0]);
//    opal_cuda_check_error(cuda_err);


#if defined (OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    buffer_size = iov[0].iov_len;
    cuda_iov_count = 1000;
    total_unpacked = 0;
    total_converted = pConvertor->bConverted;
    cuda_streams->current_stream_id = 0;
    convertor_flags = pConvertor->flags;
//    orig_stack_index = pStack->index;
    complete_flag = opal_convertor_raw( pConvertor, cuda_iov, &cuda_iov_count, &length );
    DT_CUDA_DEBUG ( opal_cuda_output(4, "Unpack complete flag %d, iov count %d, length %d, submit to CUDA stream %d\n", complete_flag, cuda_iov_count, length, cuda_streams->current_stream_id); );

#if defined (OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: ddt to iov in %ld microsec\n", total_time ); );
#endif

    dst_offset = 0;
    thread_per_block = CUDA_WARP_SIZE * 5;
    nb_blocks = 256;

    while (cuda_iov_count > 0) {

        nb_blocks_used = 0;
        cuda_iov_pipeline_block = current_cuda_device->cuda_iov_pipeline_block[iov_pipeline_block_id];
        cuda_iov_dist_h_current = cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h;
        cuda_iov_dist_d_current = cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d;
        cuda_stream_iov = cuda_iov_pipeline_block->hip_stream;
        cuda_err = hipStreamWaitEvent(*cuda_stream_iov, cuda_iov_pipeline_block->cuda_event, 0);
        opal_cuda_check_error(cuda_err);
        

#if defined (OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif

        for (i = 0; i < cuda_iov_count; i++) {
//            pElem = &(description[orig_stack_index+i]);
            if (buffer_size >= cuda_iov[i].iov_len) {
                length_per_iovec = cuda_iov[i].iov_len;
            } else {
              /*  orig_alignment = opal_datatype_basicDatatypes[pElem->elem.common.type]->size;*/
                orig_alignment = ALIGNMENT_CHAR;
                length_per_iovec = buffer_size / orig_alignment * orig_alignment;
                buffer_isfull = 1;
            }
            buffer_size -= length_per_iovec;
            total_unpacked += length_per_iovec;

            /* check alignment */
            if ((uintptr_t)(cuda_iov[i].iov_base) % ALIGNMENT_DOUBLE == 0 && (uintptr_t)source % ALIGNMENT_DOUBLE == 0 && length_per_iovec >= ALIGNMENT_DOUBLE) {
                alignment = ALIGNMENT_DOUBLE;
            } else if ((uintptr_t)(cuda_iov[i].iov_base) % ALIGNMENT_FLOAT == 0 && (uintptr_t)source % ALIGNMENT_FLOAT == 0 && length_per_iovec >= ALIGNMENT_FLOAT) {
                alignment = ALIGNMENT_FLOAT;
            } else {
                alignment = ALIGNMENT_CHAR;
            }

            //alignment = ALIGNMENT_DOUBLE;

            count_desc = length_per_iovec / alignment;
            residue_desc = length_per_iovec % alignment;
            nb_blocks_per_description = (count_desc + thread_per_block - 1) / thread_per_block;
            DT_CUDA_DEBUG ( opal_cuda_output(10, "Unpack description %d, size %d, residue %d, alignment %d\n", i, count_desc, residue_desc, alignment); );
            for (j = 0; j < nb_blocks_per_description; j++) {
                cuda_iov_dist_h_current[nb_blocks_used].dst = (unsigned char *)(cuda_iov[i].iov_base) + j * thread_per_block * alignment;
                cuda_iov_dist_h_current[nb_blocks_used].src = source;
                cuda_iov_dist_h_current[nb_blocks_used].element_alignment = alignment;
                if ( (j+1) * thread_per_block <= count_desc) {
                    cuda_iov_dist_h_current[nb_blocks_used].nb_elements = thread_per_block;// * sizeof(double);
                } else {
                    cuda_iov_dist_h_current[nb_blocks_used].nb_elements = (thread_per_block - ((j+1)*thread_per_block - count_desc));// * sizeof(double);
                }
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
                assert (cuda_iov_dist_h_current[nb_blocks_used].nb_elements > 0); 
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
                source += cuda_iov_dist_h_current[nb_blocks_used].nb_elements * alignment;
                DT_CUDA_DEBUG( opal_cuda_output(12, "Unpack \tblock %d, src %p, dst %p, nb_elements %d, alignment %d\n", nb_blocks_used, cuda_iov_dist_h_current[nb_blocks_used].src, cuda_iov_dist_h_current[nb_blocks_used].dst, cuda_iov_dist_h_current[nb_blocks_used].nb_elements, cuda_iov_dist_h_current[nb_blocks_used].element_alignment); );
                nb_blocks_used ++;
            }

            /* handle residue */
            if (residue_desc != 0) {
               /* orig_alignment = opal_datatype_basicDatatypes[pElem->elem.common.type]->size;*/
                orig_alignment = ALIGNMENT_CHAR;
                cuda_iov_dist_h_current[nb_blocks_used].dst = (unsigned char *)(cuda_iov[i].iov_base) + length_per_iovec / alignment * alignment;
                cuda_iov_dist_h_current[nb_blocks_used].src = source;
                cuda_iov_dist_h_current[nb_blocks_used].element_alignment = orig_alignment;
                cuda_iov_dist_h_current[nb_blocks_used].nb_elements = (length_per_iovec - length_per_iovec / alignment * alignment) / orig_alignment;
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
                assert (cuda_iov_dist_h_current[nb_blocks_used].nb_elements > 0);
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
                source += cuda_iov_dist_h_current[nb_blocks_used].nb_elements * orig_alignment;
                DT_CUDA_DEBUG( opal_cuda_output(12, "Unpack \tblock %d, src %p, dst %p, nb_elements %d, alignment %d\n", nb_blocks_used, cuda_iov_dist_h_current[nb_blocks_used].src, cuda_iov_dist_h_current[nb_blocks_used].dst, cuda_iov_dist_h_current[nb_blocks_used].nb_elements, cuda_iov_dist_h_current[nb_blocks_used].element_alignment); );
                nb_blocks_used ++;
            }

            if (buffer_isfull) {
                break;
            }
        }

#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: Unpack src %p, iov is prepared in %ld microsec, kernel submitted to CUDA stream %d, nb_blocks_used %d\n", source_base, total_time,  cuda_iov_pipeline_block->cuda_stream_id, nb_blocks_used); );
#endif

        hipMemcpyAsync(cuda_iov_dist_d_current, cuda_iov_dist_h_current, sizeof(ddt_cuda_iov_dist_non_cached_t)*(nb_blocks_used), hipMemcpyHostToDevice, *cuda_stream_iov);
        opal_generic_simple_unpack_cuda_iov_non_cached_kernel<<<nb_blocks, thread_per_block, 0, *cuda_stream_iov>>>(cuda_iov_dist_d_current, nb_blocks_used);
        cuda_err = hipEventRecord(cuda_iov_pipeline_block->cuda_event, *cuda_stream_iov);
        opal_cuda_check_error(cuda_err);
        iov_pipeline_block_id ++;
        iov_pipeline_block_id = iov_pipeline_block_id % NB_STREAMS;
        
        /* buffer is full */
        if (buffer_isfull) {
            size_t total_converted_tmp = total_converted;
            pConvertor->flags = convertor_flags;
            total_converted += total_unpacked;
            opal_convertor_set_position_nocheck(pConvertor, &total_converted);
            total_unpacked = total_converted - total_converted_tmp;
            break;
        }
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        convertor_flags = pConvertor->flags;
//        orig_stack_index = pStack->index;
        complete_flag = opal_convertor_raw( pConvertor, cuda_iov, &cuda_iov_count, &length );
        DT_CUDA_DEBUG ( opal_cuda_output(4, "Unpack complete flag %d, iov count %d, length %d, submit to CUDA stream %d, nb_blocks %d\n", complete_flag, cuda_iov_count, length, cuda_streams->current_stream_id, nb_blocks_used); );
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: ddt to iov in %ld microsec\n", total_time ); );
#endif

    }

    for (i = 0; i < NB_STREAMS; i++) {
        hipStreamSynchronize(cuda_streams->opal_cuda_stream[i]);
    }

    iov[0].iov_len = total_unpacked;
    *max_data = total_unpacked;
    *out_size = 1;
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack total unpacked %d\n", total_unpacked); );

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME( end_total );
    total_time = ELAPSED_TIME( start_total, end_total );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: total unpacking in %ld microsec, kernel %ld microsec\n", total_time, total_time-move_time ); );
#endif

    if( pConvertor->bConverted == pConvertor->local_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        if (pConvertor->gpu_buffer_ptr != NULL && free_required) {
            opal_ddt_cuda_free_gpu_buffer(pConvertor->gpu_buffer_ptr, 0);
            pConvertor->gpu_buffer_ptr = NULL;
        }
        return 1;
    }
    return 0;
}

#endif

int32_t opal_ddt_generic_simple_unpack_function_cuda_iov_non_cached( opal_convertor_t* pConvertor, unsigned char *source, size_t buffer_size, size_t *total_unpacked)
{
    uint32_t i;
    uint32_t nb_blocks, thread_per_block, nb_blocks_used;
    unsigned char *source_base, *destination_base;
    uint8_t buffer_isfull = 0;
    hipError_t cuda_err;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    ddt_cuda_iov_dist_cached_t* cuda_iov_dist_h_current;
    ddt_cuda_iov_dist_cached_t* cuda_iov_dist_d_current;
    ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block;
    int iov_pipeline_block_id = 0;
    hipStream_t *cuda_stream_iov = NULL;
    const struct iovec *ddt_iov = NULL;
    uint32_t ddt_iov_count = 0;
    size_t contig_disp = 0;
    uint32_t ddt_iov_start_pos, ddt_iov_end_pos, current_ddt_iov_pos;
    OPAL_PTRDIFF_TYPE ddt_extent;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif
    
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack using IOV non cached, GPU base %p, unpack from buffer %p, total size %ld\n",
                                     pConvertor->pBaseBuf, source, buffer_size); );
    
    opal_convertor_raw_cached( pConvertor, &ddt_iov, &ddt_iov_count);
    if (ddt_iov == NULL) {
        DT_CUDA_DEBUG ( opal_cuda_output(0, "Can not get ddt iov\n"););
        return OPAL_ERROR;
    }
    
  //  cuda_streams->current_stream_id = 0;
    thread_per_block = CUDA_WARP_SIZE * 5;
    nb_blocks = 256;
    source_base = source;
    opal_datatype_type_extent(pConvertor->pDesc, &ddt_extent);
    opal_ddt_set_ddt_iov_position(pConvertor, pConvertor->bConverted, ddt_iov, ddt_iov_count);
    destination_base = (unsigned char*)pConvertor->pBaseBuf + pConvertor->current_count * ddt_extent;
    
    for (i = 0; i < NB_STREAMS; i++) {
        hipStreamSynchronize(cuda_streams->ddt_cuda_stream[i]);
    }

    while( pConvertor->current_count < pConvertor->count && !buffer_isfull) {

        nb_blocks_used = 0;
        ddt_iov_start_pos = pConvertor->current_iov_pos;
        ddt_iov_end_pos = ddt_iov_start_pos + IOV_PIPELINE_SIZE;
        if (ddt_iov_end_pos > ddt_iov_count) {
            ddt_iov_end_pos = ddt_iov_count;
        }
        cuda_iov_pipeline_block = current_cuda_device->cuda_iov_pipeline_block[iov_pipeline_block_id];
        cuda_iov_pipeline_block->hip_stream = &(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
        cuda_iov_dist_h_current = cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h;
        cuda_iov_dist_d_current = cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d;
        cuda_stream_iov = cuda_iov_pipeline_block->hip_stream;
        cuda_err = hipEventSynchronize(cuda_iov_pipeline_block->cuda_event);
        opal_cuda_check_error(cuda_err);
        

#if defined (OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif

        buffer_isfull = opal_ddt_iov_to_cuda_iov(pConvertor, ddt_iov, cuda_iov_dist_h_current, ddt_iov_start_pos, ddt_iov_end_pos, &buffer_size, &nb_blocks_used, total_unpacked, &contig_disp, &current_ddt_iov_pos);

#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: Unpack src %p to dest %p, iov is prepared in %ld microsec, kernel submitted to CUDA stream %d, nb_blocks_used %d\n", source_base, destination_base, total_time,  cuda_streams->current_stream_id, nb_blocks_used); );
#endif

        hipMemcpyAsync(cuda_iov_dist_d_current, cuda_iov_dist_h_current, sizeof(ddt_cuda_iov_dist_cached_t)*(nb_blocks_used+1), hipMemcpyHostToDevice, *cuda_stream_iov);
        opal_generic_simple_unpack_cuda_iov_cached_kernel<<<nb_blocks, thread_per_block, 0, *cuda_stream_iov>>>(cuda_iov_dist_d_current, 0, nb_blocks_used, 0, 0, nb_blocks_used, destination_base, source_base, 0, 0);
        //hipStreamSynchronize(*cuda_stream_iov);
        cuda_err = hipEventRecord(cuda_iov_pipeline_block->cuda_event, *cuda_stream_iov);
        opal_cuda_check_error(cuda_err);
        iov_pipeline_block_id ++;
        iov_pipeline_block_id = iov_pipeline_block_id % NB_STREAMS;
        source_base += contig_disp;
        if (!buffer_isfull) {
            pConvertor->current_iov_pos = current_ddt_iov_pos;
            if (current_ddt_iov_pos == ddt_iov_count) {
                pConvertor->current_count ++;
                pConvertor->current_iov_pos = 0;
                destination_base += ddt_extent;
            }
        }
    }

    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);

    return OPAL_SUCCESS;
}

int32_t opal_ddt_generic_simple_unpack_function_cuda_iov_cached( opal_convertor_t* pConvertor, unsigned char *source, size_t buffer_size, size_t *total_unpacked)
{
    uint32_t i;
    uint32_t nb_blocks, thread_per_block, nb_blocks_used;
    unsigned char *source_base, *destination_base;
    uint8_t buffer_isfull = 0;
    hipError_t cuda_err;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
    hipStream_t *cuda_stream_iov = NULL;
    uint32_t cuda_iov_start_pos, cuda_iov_end_pos;
    ddt_cuda_iov_total_cached_t* cached_cuda_iov = NULL;
    ddt_cuda_iov_dist_cached_t* cached_cuda_iov_dist_d = NULL;
    uint32_t *cached_cuda_iov_nb_bytes_list_h = NULL;
    uint32_t cached_cuda_iov_count = 0;
    size_t cuda_iov_partial_length_start = 0;
    size_t cuda_iov_partial_length_end = 0;
    opal_datatype_count_t convertor_current_count;
    OPAL_PTRDIFF_TYPE ddt_extent;

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif
    
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack using IOV cached, GPU base %p, unpack from buffer %p, total size %ld\n",
                                     pConvertor->pBaseBuf, source, buffer_size); );

#if defined (OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif

 //   cuda_streams->current_stream_id = 0;
    source_base = source;
    thread_per_block = CUDA_WARP_SIZE * 8;
    nb_blocks = 64;
    destination_base = (unsigned char*)pConvertor->pBaseBuf;
    
    /* cuda iov is not cached, start to cache iov */
    if(opal_ddt_cuda_iov_is_cached(pConvertor) == 0) {
#if defined (OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        if (opal_ddt_cache_cuda_iov(pConvertor, &nb_blocks_used) == OPAL_SUCCESS) {
            opal_ddt_set_cuda_iov_cached(pConvertor, nb_blocks_used);
            DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack cuda iov is cached, count %d\n", nb_blocks_used););
        }
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: Unpack cuda iov is cached in %ld microsec, nb_blocks_used %d\n", total_time, nb_blocks_used); );
#endif
    }
      
    /* now we use cached cuda iov */
    opal_ddt_get_cached_cuda_iov(pConvertor, &cached_cuda_iov);
    cached_cuda_iov_dist_d = cached_cuda_iov->cuda_iov_dist_d;
    assert(cached_cuda_iov_dist_d != NULL);
    cached_cuda_iov_nb_bytes_list_h = cached_cuda_iov->nb_bytes_h;
    assert(cached_cuda_iov_nb_bytes_list_h != NULL);
    
    cached_cuda_iov_count = cached_cuda_iov->cuda_iov_count;
    opal_ddt_set_cuda_iov_position(pConvertor, pConvertor->bConverted, cached_cuda_iov_nb_bytes_list_h, cached_cuda_iov_count);
    cuda_iov_start_pos = pConvertor->current_cuda_iov_pos;
    cuda_iov_end_pos = cached_cuda_iov_count;
    nb_blocks_used = 0;
    cuda_iov_pipeline_block = current_cuda_device->cuda_iov_pipeline_block[0];
    cuda_iov_pipeline_block->hip_stream = &(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
    cuda_stream_iov = cuda_iov_pipeline_block->hip_stream;
    convertor_current_count = pConvertor->current_count;
    
    if (pConvertor->current_iov_partial_length > 0) {
        cuda_iov_partial_length_start = pConvertor->current_iov_partial_length;
        *total_unpacked += cuda_iov_partial_length_start;
        buffer_size -= cuda_iov_partial_length_start;
        pConvertor->current_iov_partial_length = 0;
        cuda_iov_start_pos ++;
        nb_blocks_used ++;
    }
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    while( pConvertor->current_count < pConvertor->count && !buffer_isfull) {
        for (i = cuda_iov_start_pos; i < cuda_iov_end_pos && !buffer_isfull; i++) {
            if (buffer_size >= cached_cuda_iov_nb_bytes_list_h[i]) {
                *total_unpacked += cached_cuda_iov_nb_bytes_list_h[i];
                buffer_size -= cached_cuda_iov_nb_bytes_list_h[i];
                nb_blocks_used ++;
            } else {
                if (buffer_size > 0) {
                    cuda_iov_partial_length_end = buffer_size;
                    *total_unpacked += cuda_iov_partial_length_end;
                    nb_blocks_used ++;
                }
                buffer_size = 0;
                buffer_isfull = 1;
                break;
            }
        }
        if (!buffer_isfull) {
            pConvertor->current_count ++;
            cuda_iov_start_pos = 0;
            cuda_iov_end_pos = cached_cuda_iov_count;
        }
    }
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: Unpack src %p, cached cuda iov is prepared in %ld microsec, kernel submitted to CUDA stream %d, nb_blocks %d\n", source_base, total_time,  cuda_streams->current_stream_id, nb_blocks_used); );
#endif
    opal_datatype_type_extent(pConvertor->pDesc, &ddt_extent);
    DT_CUDA_DEBUG ( opal_cuda_output(2, "Unpack kernel launched src_base %p, dst_base %p, nb_blocks %ld\n", source_base, destination_base, nb_blocks_used ); );

 //   hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif    
    opal_generic_simple_unpack_cuda_iov_cached_kernel<<<nb_blocks, thread_per_block, 0, *cuda_stream_iov>>>(cached_cuda_iov_dist_d, pConvertor->current_cuda_iov_pos, cached_cuda_iov_count, ddt_extent, convertor_current_count, nb_blocks_used, destination_base, source_base, cuda_iov_partial_length_start, cuda_iov_partial_length_end);

    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG ( opal_cuda_output(2, "[Timing]: Unpack kernel %ld microsec\n", total_time); );
#endif

    return OPAL_SUCCESS;
}

void unpack_contiguous_loop_cuda( dt_elem_desc_t* ELEM,
                                  uint32_t* COUNT,
                                  unsigned char** SOURCE,
                                  unsigned char** DESTINATION,
                                  size_t* SPACE )
{
    ddt_loop_desc_t *_loop = (ddt_loop_desc_t*)(ELEM);
    ddt_endloop_desc_t* _end_loop = (ddt_endloop_desc_t*)((ELEM) + _loop->items);
    unsigned char* _destination = (*DESTINATION) + _end_loop->first_elem_disp;
    uint32_t _copy_loops = *(COUNT);
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _source = *(SOURCE);
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time;
#endif
    
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Unpack using contiguous_loop_cuda\n"); );

    if( (_copy_loops * _end_loop->size) > *(SPACE) )
        _copy_loops = (uint32_t)(*(SPACE) / _end_loop->size);

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
//    tasks_per_block = THREAD_PER_BLOCK * TASK_PER_THREAD;
//    num_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;
#if OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_AS_KERNEL
     hipMemcpy2DAsync(_destination, _loop->extent, _source, _end_loop->size, _end_loop->size, _copy_loops, hipMemcpyDeviceToDevice, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
#else
     unpack_contiguous_loop_cuda_kernel_global<<<192, 4*THREAD_PER_BLOCK, 0, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]>>>(_copy_loops, _end_loop->size, _loop->extent, _source, _destination);
#endif /* OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_AS_KERNEL */

#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    *(DESTINATION) = _destination + _loop->extent*_copy_loops - _end_loop->first_elem_disp;
    *(SOURCE) = *(SOURCE)  + _copy_loops * _end_loop->size;
    *(SPACE) -= _copy_loops * _end_loop->size;
    *(COUNT) -= _copy_loops;
#endif

    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
#if defined(OPAL_DATATYPE_CUDA_TIMING) 
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG( opal_cuda_output( 2, "[Timing]: vector unpacking in %ld microsec\n", total_time ); );
#endif
}

void unpack_contiguous_loop_cuda_memcpy2d( dt_elem_desc_t* ELEM,
                                  uint32_t* COUNT,
                                  unsigned char** SOURCE,
                                  unsigned char** DESTINATION,
                                  size_t* SPACE )
{
    ddt_loop_desc_t *_loop = (ddt_loop_desc_t*)(ELEM);
    ddt_endloop_desc_t* _end_loop = (ddt_endloop_desc_t*)((ELEM) + _loop->items);
    unsigned char* _destination = (*DESTINATION) + _end_loop->first_elem_disp;
    uint32_t _copy_loops = *(COUNT);
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _source = *(SOURCE);
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time;
#endif
    
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Unpack using contiguous_loop_cuda_memcpy2d\n"); );

    if( (_copy_loops * _end_loop->size) > *(SPACE) )
        _copy_loops = (uint32_t)(*(SPACE) / _end_loop->size);

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    hipMemcpy2DAsync(_destination, _loop->extent, _source, _end_loop->size, _end_loop->size, _copy_loops, hipMemcpyHostToDevice, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);

#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    *(DESTINATION) = _destination + _loop->extent*_copy_loops - _end_loop->first_elem_disp;
    *(SOURCE) = *(SOURCE)  + _copy_loops * _end_loop->size;
    *(SPACE) -= _copy_loops * _end_loop->size;
    *(COUNT) -= _copy_loops;
#endif
    
    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
    
#if defined(OPAL_DATATYPE_CUDA_TIMING) 
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG( opal_cuda_output( 2, "[Timing]: vector unpacking with memcpy2d in %ld microsec\n", total_time ); );
#endif
}

void unpack_contiguous_loop_cuda_zerocopy( dt_elem_desc_t* ELEM,
                                           uint32_t* COUNT,
                                           unsigned char** SOURCE,
                                           unsigned char** DESTINATION,
                                           size_t* SPACE)
{
    ddt_loop_desc_t *_loop = (ddt_loop_desc_t*)(ELEM);
    ddt_endloop_desc_t* _end_loop = (ddt_endloop_desc_t*)((ELEM) + _loop->items);
    unsigned char* _destination = (*DESTINATION) + _end_loop->first_elem_disp;
    uint32_t _copy_loops = *(COUNT);
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _source = *(SOURCE);
    unsigned char* _source_dev;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time;
#endif
    
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Unpack using contiguous_loop_cuda_zerocopy\n"); );

    if( (_copy_loops * _end_loop->size) > *(SPACE) )
        _copy_loops = (uint32_t)(*(SPACE) / _end_loop->size);

#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
//    tasks_per_block = THREAD_PER_BLOCK * TASK_PER_THREAD;
//    num_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;

    hipError_t reg_rv = hipHostGetDevicePointer((void **)&_source_dev, (void *) _source, 0);
    if (reg_rv != hipSuccess) {
        const char *cuda_err = hipGetErrorString(reg_rv);
        printf("can not get dev mem, %s\n", cuda_err);
    }
#if OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_AS_KERNEL
    hipMemcpy2DAsync(_destination, _loop->extent, _source_dev, _end_loop->size, _end_loop->size, _copy_loops, hipMemcpyDeviceToDevice, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
#else
    unpack_contiguous_loop_cuda_kernel_global<<<192, 4*THREAD_PER_BLOCK, 0, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]>>>(_copy_loops, _end_loop->size, _loop->extent, _source_dev, _destination);
#endif /* OPAL_DATATYPE_VECTOR_USE_MEMCPY2D_AS_KERNEL */

#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    *(DESTINATION) = _destination + _loop->extent*_copy_loops - _end_loop->first_elem_disp;
    *(SOURCE) = *(SOURCE)  + _copy_loops * _end_loop->size;
    *(SPACE) -= _copy_loops * _end_loop->size;
    *(COUNT) -= _copy_loops;
#endif

    hipStreamSynchronize(cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]);
  //  hipHostUnregister(_source);
#if defined(OPAL_DATATYPE_CUDA_TIMING) 
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    DT_CUDA_DEBUG( opal_cuda_output( 2, "[Timing]: vector unpacking in %ld microsec\n", total_time ); );
#endif
}

void unpack_predefined_data_cuda( dt_elem_desc_t* ELEM,
                                  uint32_t* COUNT,
                                  unsigned char** SOURCE,
                                  unsigned char** DESTINATION,
                                  size_t* SPACE )
{
    uint32_t _copy_count = *(COUNT);
    size_t _copy_blength;
    ddt_elem_desc_t* _elem = &((ELEM)->elem);
    unsigned char* _source = (*SOURCE);
    uint32_t nb_blocks, tasks_per_block, thread_per_block;
    unsigned char* _destination = *(DESTINATION) + _elem->disp;
    
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;

    _copy_blength = 8;//opal_datatype_basicDatatypes[_elem->common.type]->size;
    if( (_copy_count * _copy_blength) > *(SPACE) ) {
        _copy_count = (uint32_t)(*(SPACE) / _copy_blength);
        if( 0 == _copy_count ) return;  /* nothing to do */
    }
    
    
    if (*COUNT / TASK_PER_THREAD < CUDA_WARP_SIZE) {
        thread_per_block = CUDA_WARP_SIZE;
    } else if (*COUNT / TASK_PER_THREAD < CUDA_WARP_SIZE * 2) {
        thread_per_block = CUDA_WARP_SIZE * 2;
    } else if (*COUNT / TASK_PER_THREAD < CUDA_WARP_SIZE * 3) {
        thread_per_block = CUDA_WARP_SIZE * 3;
    } else {
        thread_per_block = CUDA_WARP_SIZE * 5;
    }
    tasks_per_block = thread_per_block * TASK_PER_THREAD;
    nb_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;

 //   DBGPRINT("num_blocks %d, thread %d\n", nb_blocks, tasks_per_block);
 //   DBGPRINT( "GPU pack 1. memcpy( %p, %p, %lu ) => space %lu\n", _destination, _source, (unsigned long)_copy_count, (unsigned long)(*(SPACE)) );
    
    unpack_contiguous_loop_cuda_kernel_global<<<nb_blocks, thread_per_block, 0, cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id]>>>(_copy_count, _copy_blength, _elem->extent, _source, _destination);
    cuda_streams->current_stream_id ++;
    cuda_streams->current_stream_id = cuda_streams->current_stream_id % NB_STREAMS;
    
#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)  
    _copy_blength *= _copy_count;
    *(DESTINATION)  = _destination + _elem->extent*_copy_count - _elem->disp;
    *(SOURCE) += _copy_blength;
    *(SPACE)  -= _copy_blength;
    *(COUNT)  -= _copy_count;
#endif
    
}
