#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdarg.h> 

/*
 * NOTE: The order of this array *MUST* match what is listed in datatype.h
 * (use of designated initializers should relax this restrictions some)
 */
/*
OPAL_DECLSPEC const size_t opal_datatype_basicDatatypesSize[OPAL_DATATYPE_MAX_PREDEFINED] = {
    OPAL_DATATYPE_LOOP_SIZE,
    OPAL_DATATYPE_END_LOOP_SIZE,
    OPAL_DATATYPE_LB_SIZE,
    OPAL_DATATYPE_UB_SIZE,
    OPAL_DATATYPE_INT1_SIZE,
    OPAL_DATATYPE_INT2_SIZE,
    OPAL_DATATYPE_INT4_SIZE,
    OPAL_DATATYPE_INT8_SIZE,
    OPAL_DATATYPE_INT16_SIZE,   
    OPAL_DATATYPE_UINT1_SIZE,
    OPAL_DATATYPE_UINT2_SIZE,
    OPAL_DATATYPE_UINT4_SIZE,
    OPAL_DATATYPE_UINT8_SIZE,
    OPAL_DATATYPE_UINT16_SIZE,  
    OPAL_DATATYPE_FLOAT2_SIZE,
    OPAL_DATATYPE_FLOAT4_SIZE,
    OPAL_DATATYPE_FLOAT8_SIZE,
    OPAL_DATATYPE_FLOAT12_SIZE,
    OPAL_DATATYPE_FLOAT16_SIZE,
    OPAL_DATATYPE_FLOAT_COMPLEX_SIZE,
    OPAL_DATATYPE_DOUBLE_COMPLEX_SIZE,
    OPAL_DATATYPE_LONG_DOUBLE_COMPLEX_SIZE,
    OPAL_DATATYPE_BOOL_SIZE,
    OPAL_DATATYPE_WCHAR_SIZE,
    OPAL_DATATYPE_UNAVAILABLE_SIZE,
};
*/
/***** my variables ********/


ddt_cuda_list_t *cuda_free_list;
ddt_cuda_device_t *cuda_device;
ddt_cuda_desc_t *cuda_desc_d, *cuda_desc_h;
unsigned char *pBaseBuf_GPU, *gpu_src_const, *gpu_dest_const;
unsigned char *ddt_cuda_pack_buffer, *ddt_cuda_unpack_buffer;
ddt_cuda_stream_t* cuda_streams;
struct iovec cuda_iov[CUDA_NB_IOV];
uint32_t cuda_iov_count;
ddt_cuda_description_dist_t description_dist_h[CUDA_MAX_NB_BLOCKS];
ddt_cuda_description_dist_t* description_dist_d;
ddt_cuda_iov_dist_t cuda_iov_dist_h[NB_STREAMS][CUDA_MAX_NB_BLOCKS];
ddt_cuda_iov_dist_t* cuda_iov_dist_d[NB_STREAMS];
dt_elem_desc_t* description_d;
uint8_t opal_datatype_cuda_debug;

//uint8_t ALIGNMENT_DOUBLE, ALIGNMENT_FLOAT, ALIGNMENT_CHAR;


static inline ddt_cuda_buffer_t* obj_ddt_cuda_buffer_new()
{
    ddt_cuda_buffer_t *p = (ddt_cuda_buffer_t *)malloc(sizeof(ddt_cuda_buffer_t));
    p->next = NULL;
    p->prev = NULL;
    p->size = 0;
    p->gpu_addr = NULL;
    return p; 
}

static inline void obj_ddt_cuda_buffer_chop(ddt_cuda_buffer_t *p)
{
    p->next = NULL;
    p->prev = NULL;
}

static inline void obj_ddt_cuda_buffer_reset(ddt_cuda_buffer_t *p)
{
    p->size = 0;
    p->gpu_addr = NULL;
}

static ddt_cuda_list_t* init_cuda_free_list()
{
    ddt_cuda_list_t *list = NULL;
    ddt_cuda_buffer_t *p, *prev;
    int i;
    list = (ddt_cuda_list_t *)malloc(sizeof(ddt_cuda_list_t));
    p = obj_ddt_cuda_buffer_new();
    list->head = p;
    prev = p;
    for (i = 1; i < DT_CUDA_FREE_LIST_SIZE; i++) {
        p = obj_ddt_cuda_buffer_new();
        prev->next = p;
        p->prev = prev;
        prev = p;
    }
    list->tail = p;
    list->nb_elements = DT_CUDA_FREE_LIST_SIZE;
    return list;
} 

static inline ddt_cuda_buffer_t* cuda_list_pop_tail(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *p = NULL;
    p = list->tail;
    if (p == NULL) {
        return p;
    } else {
        list->nb_elements --;
        if (list->head == p) {
            list->head = NULL;
            list->tail = NULL;
        } else {
            list->tail = p->prev;
            p->prev->next = NULL;
            obj_ddt_cuda_buffer_chop(p);
        }
        return p;
    }
}

static inline void cuda_list_push_head(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_head = list->head;
    assert(item->next == NULL && item->prev == NULL);
    list->head = item;
    item->next = orig_head;
    if (orig_head == NULL) {
        list->tail = item;
    } else {
        orig_head->prev = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_push_tail(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_tail = list->tail;
    assert(item->next == NULL && item->prev == NULL);
    list->tail = item;
    item->prev = orig_tail;
    if (orig_tail == NULL) {
        list->head = item;
    } else {
        orig_tail->next = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_delete(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    if (item->prev == NULL && item->next == NULL) {
        list->head = NULL;
        list->tail = NULL;
    }else if (item->prev == NULL && item->next != NULL) {
        list->head = item->next;
        item->next->prev = NULL;
    } else if (item->next == NULL && item->prev != NULL) {
        list->tail = item->prev;
        item->prev->next = NULL;
    } else {
        item->prev->next = item->next;
        item->next->prev = item->prev;
    }
    list->nb_elements --;
    obj_ddt_cuda_buffer_chop(item);
}

static inline void cuda_list_insert_before(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item, ddt_cuda_buffer_t *next)
{
    assert(item->next == NULL && item->prev == NULL);
    item->next = next;
    item->prev = next->prev;
    next->prev = item;
    if (list->head == next) {
        list->head = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_item_merge_by_addr(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *ptr = NULL;
    ddt_cuda_buffer_t *next = NULL;
    ptr = list->head;
    while(ptr != NULL) {
        next = ptr->next;
        if (next == NULL) {
            break;
        } else if ((ptr->gpu_addr + ptr->size) == next->gpu_addr) {
            ptr->size += next->size;
            cuda_list_delete(list, next);
        } else {
            ptr = ptr->next;
        }
    }
}

void opal_datatype_cuda_init(void)
{
    uint32_t i;
    int device;
    hipError_t res;

    res = hipGetDevice(&device);
    if( hipSuccess != res ) {
        opal_cuda_output(0, "Cannot retrieve the device being used. Drop CUDA support!\n");
        return;
    }    
    printf("current device %d\n", device);

    cuda_free_list = init_cuda_free_list();
    
    /* init device */
    cuda_device = (ddt_cuda_device_t *)malloc(sizeof(ddt_cuda_device_t)*1);
    for (i = 0; i < 1; i++) {
        unsigned char *gpu_ptr = NULL;
        if (hipMalloc((void **)(&gpu_ptr), sizeof(char)*DT_CUDA_BUFFER_SIZE) != hipSuccess) {
            DT_CUDA_DEBUG( opal_cuda_output( 0, "hipMalloc is failed in GPU %d\n", i); );
        }
        hipMemset(gpu_ptr, 0, sizeof(char)*DT_CUDA_BUFFER_SIZE);
        cuda_device[i].gpu_buffer = gpu_ptr;
        
        cuda_device[i].buffer_free_size = DT_CUDA_BUFFER_SIZE;
        ddt_cuda_buffer_t *p = obj_ddt_cuda_buffer_new();
        p->size = DT_CUDA_BUFFER_SIZE;
        p->gpu_addr = gpu_ptr;
        cuda_device[i].buffer_free.head = p;
        cuda_device[i].buffer_free.tail = cuda_device[i].buffer_free.head;
        cuda_device[i].buffer_free.nb_elements = 1;
        
        cuda_device[i].buffer_used.head = NULL;
        cuda_device[i].buffer_used.tail = NULL;
        cuda_device[i].buffer_used_size = 0;
        cuda_device[i].buffer_used.nb_elements = 0;
    }
    
    hipMalloc((void **)&cuda_desc_d, sizeof(ddt_cuda_desc_t));
    hipHostMalloc((void **)&cuda_desc_h, sizeof(ddt_cuda_desc_t));
    printf("size cuda_desc %d\n", sizeof(ddt_cuda_desc_t));
    
    // printf("malloc iov\n");
    // for (i = 0; i < IOV_ARRAY_SIZE; i++) {
    //     void* iov_base;
    //     hipMalloc( (void **)&iov_base, sizeof(char)*IOV_LEN);
    //     cuda_desc_h->iov[i].iov_base = iov_base;
    //     cuda_desc_h->iov[i].iov_len = IOV_LEN;
    // }
    
    hipMalloc((void **)(&ddt_cuda_pack_buffer), sizeof(char)*DT_CUDA_BUFFER_SIZE);
    printf("malloc cuda packing buffer, %p\n", ddt_cuda_pack_buffer);
    hipMalloc((void **)(&ddt_cuda_unpack_buffer), sizeof(char)*DT_CUDA_BUFFER_SIZE);
    printf("malloc cuda unpacking buffer, %p\n", ddt_cuda_unpack_buffer);

    cuda_desc_h->iov[0].iov_base = ddt_cuda_pack_buffer;
    cuda_desc_h->iov[0].iov_len = DT_CUDA_BUFFER_SIZE;
    
    hipMalloc((void **)(&pBaseBuf_GPU), sizeof(char)*DT_CUDA_BUFFER_SIZE);
    gpu_src_const = pBaseBuf_GPU;
    gpu_dest_const = (unsigned char*)cuda_desc_h->iov[0].iov_base; 
    
    cuda_desc_h->description_max_count = 0;
    cuda_desc_h->description_count = 0;
    
    /* init cuda stream */
    cuda_streams = (ddt_cuda_stream_t*)malloc(sizeof(ddt_cuda_stream_t));
    for (i = 0; i < NB_STREAMS; i++) {
        hipStreamCreate(&(cuda_streams->opal_cuda_stream[i]));
    }
    cuda_streams->current_stream_id = 0;
    
    /* init cuda_iov */
    cuda_iov_count = CUDA_NB_IOV;
    
    /* init description dist array */
    hipMalloc((void **)(&description_dist_d), sizeof(ddt_cuda_description_dist_t)*CUDA_MAX_NB_BLOCKS);
    cuda_desc_h->description_dist = description_dist_d;
    
    /* only for iov version */
    for (i = 0; i < NB_STREAMS; i++) {
        hipMalloc((void **)(&cuda_iov_dist_d[i]), sizeof(ddt_cuda_iov_dist_t)*CUDA_MAX_NB_BLOCKS);
    }
    
    opal_datatype_cuda_debug = 1;
    
    // /* init size for double, float, char */
    // ALIGNMENT_DOUBLE = sizeof(double);
    // ALIGNMENT_FLOAT = sizeof(float);
    // ALIGNMENT_CHAR = sizeof(char);
    
    hipDeviceSynchronize();
}

void opal_datatype_cuda_fini(void)
{
    uint32_t i;
    
    if (cuda_desc_d != NULL) {
        hipFree(cuda_desc_d);
        cuda_desc_d = NULL;
    }
    if (cuda_desc_h->description != NULL) {
        hipFree(cuda_desc_h->description);
        cuda_desc_h->description = NULL;
    }
    if (cuda_desc_h->description_dist != NULL) {
        hipFree(cuda_desc_h->description_dist);
        cuda_desc_h->description_dist = NULL;
    }
    printf("free iov\n");
    if (cuda_desc_h != NULL) {    
        for (i = 0; i < IOV_ARRAY_SIZE; i++) {
            hipFree(cuda_desc_h->iov[i].iov_base);
            cuda_desc_h->iov[i].iov_base = NULL;
        }
    
        hipHostFree(cuda_desc_h);
        cuda_desc_h = NULL;
    }
    
    /* destory cuda stream */
    for (i = 0; i < NB_STREAMS; i++) {
        hipStreamDestroy(cuda_streams->opal_cuda_stream[i]);
    }
    free(cuda_streams);
    
    /* only for iov version */
    for (i = 0; i < NB_STREAMS; i++) {
        hipFree(cuda_iov_dist_d[i]);
    }
}

void opal_cuda_sync_device(void)
{
    hipDeviceSynchronize();
    pBaseBuf_GPU = gpu_src_const;
    cuda_desc_h->iov[0].iov_base = (void*)gpu_dest_const;
}

int32_t opal_cuda_is_gpu_buffer(const void *ptr)
{
    int res;
    hipMemoryType memType;
    hipDeviceptr_t dbuf = (hipDeviceptr_t)ptr;
    res = hipPointerGetAttribute(&memType, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dbuf);
    if (res != hipSuccess) {
        /* If we cannot determine it is device pointer,
         * just assume it is not. */
        printf("!!!!!!! %p is not a gpu buffer. Take no-CUDA path!\n", ptr);
        return 0;
    }
    /* Anything but hipMemoryTypeDevice is not a GPU memory */
    return (memType == hipMemoryTypeDevice) ? 1 : 0;
}

unsigned char* opal_cuda_get_gpu_pack_buffer()
{
    if (ddt_cuda_pack_buffer != NULL) {
        return ddt_cuda_pack_buffer;
    } else {
        return NULL;
    }
}

void* opal_cuda_malloc_gpu_buffer(size_t size, int gpu_id)
{
    int dev_id;
    hipGetDevice(&dev_id);
    ddt_cuda_device_t *device = &cuda_device[gpu_id];
    if (device->buffer_free_size < size) {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "No GPU buffer at dev_id %d.\n", dev_id); );
        return NULL;
    }
    ddt_cuda_buffer_t *ptr = NULL;
    void *addr = NULL;
    ptr = device->buffer_free.head;
    while (ptr != NULL) {
        if (ptr->size >= size) {
            addr = ptr->gpu_addr;
            ptr->size -= size;
            if (ptr->size == 0) {
                cuda_list_delete(&device->buffer_free, ptr);
                obj_ddt_cuda_buffer_reset(ptr);
                cuda_list_push_head(cuda_free_list, ptr);
            } else {
                ptr->gpu_addr += size;
            }
            break;
        }
        ptr = ptr->next;
    }
    
    if (ptr == NULL) {
        return NULL;
    } else {    
        ddt_cuda_buffer_t *p = cuda_list_pop_tail(cuda_free_list);
        if (p == NULL) {
            p = obj_ddt_cuda_buffer_new();
        }
        p->size = size;
        p->gpu_addr = (unsigned char*)addr;
        cuda_list_push_head(&device->buffer_used, p);
        device->buffer_used_size += size;
        device->buffer_free_size -= size;
        DT_CUDA_DEBUG( opal_cuda_output( 0, "Malloc GPU buffer %p, dev_id %d.\n", addr, dev_id); );
        return addr;
    }
}

void opal_cuda_free_gpu_buffer(void *addr, int gpu_id)
{
    ddt_cuda_device_t *device = &cuda_device[gpu_id];
    ddt_cuda_buffer_t *ptr = NULL;
    ddt_cuda_buffer_t *ptr_next = NULL;
    ptr = device->buffer_used.head;
    while (ptr != NULL) {
        if (ptr->gpu_addr == addr) {
            cuda_list_delete(&device->buffer_used, ptr);
            ptr_next = device->buffer_free.head;
            while (ptr_next != NULL) {
                if (ptr_next->gpu_addr > addr) {
                    break;
                }
                ptr_next = ptr_next->next;
            }
            if (ptr_next == NULL) {
                /* buffer_free is empty, or insert to last one */
                cuda_list_push_tail(&device->buffer_free, ptr);
            } else {
                cuda_list_insert_before(&device->buffer_free, ptr, ptr_next);
            }
            cuda_list_item_merge_by_addr(&device->buffer_free);
            device->buffer_free_size += ptr->size;
            break;
        }
        ptr = ptr->next;
    }
    if (ptr == NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "addr %p is not managed.\n", addr); );
    }
    size_t size = ptr->size;
    cuda_list_item_merge_by_addr(&device->buffer_free, ptr);
    device->buffer_free_size += size;
    device->buffer_used_size -= size;
    DT_CUDA_DEBUG( opal_cuda_output( 0, "Free GPU buffer %p.\n", addr); );
}

void opal_dump_cuda_list(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *ptr = NULL;
    ptr = list->head;
    DT_CUDA_DEBUG( opal_cuda_output( 0, "DUMP cuda list %p, nb_elements %d\n", list, list->nb_elements); );
    while (ptr != NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "\titem addr %p, size %ld.\n", ptr->gpu_addr, ptr->size); );
        ptr = ptr->next;
    }
}

/* from internal.h*/
void opal_cuda_output(int output_id, const char *format, ...)
{
    if (output_id >= 0 && output_id <= OPAL_DATATYPE_CUDA_DEBUG_LEVEL) {
        va_list arglist;
        fprintf( stderr, "[Debug %d]: ", output_id );
        va_start(arglist, format);
        vfprintf(stderr, format, arglist);
        va_end(arglist);
    }
}
