#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdarg.h> 


ddt_cuda_list_t *cuda_free_list;
ddt_cuda_device_t *cuda_devices;
ddt_cuda_device_t *current_cuda_device;
struct iovec cuda_iov[CUDA_NB_IOV];
uint32_t cuda_iov_count;

//uint8_t ALIGNMENT_DOUBLE, ALIGNMENT_FLOAT, ALIGNMENT_CHAR;


static inline ddt_cuda_buffer_t* obj_ddt_cuda_buffer_new()
{
    ddt_cuda_buffer_t *p = (ddt_cuda_buffer_t *)malloc(sizeof(ddt_cuda_buffer_t));
    p->next = NULL;
    p->prev = NULL;
    p->size = 0;
    p->gpu_addr = NULL;
    return p; 
}

static inline void obj_ddt_cuda_buffer_chop(ddt_cuda_buffer_t *p)
{
    p->next = NULL;
    p->prev = NULL;
}

static inline void obj_ddt_cuda_buffer_reset(ddt_cuda_buffer_t *p)
{
    p->size = 0;
    p->gpu_addr = NULL;
}

static ddt_cuda_list_t* init_cuda_free_list()
{
    ddt_cuda_list_t *list = NULL;
    ddt_cuda_buffer_t *p, *prev;
    int i;
    list = (ddt_cuda_list_t *)malloc(sizeof(ddt_cuda_list_t));
    p = obj_ddt_cuda_buffer_new();
    list->head = p;
    prev = p;
    for (i = 1; i < DT_CUDA_FREE_LIST_SIZE; i++) {
        p = obj_ddt_cuda_buffer_new();
        prev->next = p;
        p->prev = prev;
        prev = p;
    }
    list->tail = p;
    list->nb_elements = DT_CUDA_FREE_LIST_SIZE;
    return list;
} 

static inline ddt_cuda_buffer_t* cuda_list_pop_tail(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *p = NULL;
    p = list->tail;
    if (p == NULL) {
        return p;
    } else {
        list->nb_elements --;
        if (list->head == p) {
            list->head = NULL;
            list->tail = NULL;
        } else {
            list->tail = p->prev;
            p->prev->next = NULL;
            obj_ddt_cuda_buffer_chop(p);
        }
        return p;
    }
}

static inline void cuda_list_push_head(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_head = list->head;
    assert(item->next == NULL && item->prev == NULL);
    list->head = item;
    item->next = orig_head;
    if (orig_head == NULL) {
        list->tail = item;
    } else {
        orig_head->prev = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_push_tail(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_tail = list->tail;
    assert(item->next == NULL && item->prev == NULL);
    list->tail = item;
    item->prev = orig_tail;
    if (orig_tail == NULL) {
        list->head = item;
    } else {
        orig_tail->next = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_delete(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    if (item->prev == NULL && item->next == NULL) {
        list->head = NULL;
        list->tail = NULL;
    }else if (item->prev == NULL && item->next != NULL) {
        list->head = item->next;
        item->next->prev = NULL;
    } else if (item->next == NULL && item->prev != NULL) {
        list->tail = item->prev;
        item->prev->next = NULL;
    } else {
        item->prev->next = item->next;
        item->next->prev = item->prev;
    }
    list->nb_elements --;
    obj_ddt_cuda_buffer_chop(item);
}

static inline void cuda_list_insert_before(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item, ddt_cuda_buffer_t *next)
{
    assert(item->next == NULL && item->prev == NULL);
    item->next = next;
    item->prev = next->prev;
    next->prev = item;
    if (list->head == next) {
        list->head = item;
    }
    list->nb_elements ++;
}

/**
 * Collapse the list of free buffers by mergining consecutive buffers. As the property of this list
 * is continously maintained, we only have to parse it up to the newest inserted elements.
 */
static inline void cuda_list_item_merge_by_addr(ddt_cuda_list_t *list, ddt_cuda_buffer_t* last)
{
    ddt_cuda_buffer_t *current = list->head;
    ddt_cuda_buffer_t *next = NULL;
    void* stop_addr = last->gpu_addr;

    while(1) {  /* loop forever, the exit conditions are inside */
        if( NULL == (next = current->next) ) return;
        if ((current->gpu_addr + current->size) == next->gpu_addr) {
            current->size += next->size;
            cuda_list_delete(list, next);
            free(next);  /* release the element, and try to continue merging */
            continue;
        }
        current = current->next;
        if( NULL == current ) return;
        if( current->gpu_addr > stop_addr ) return;
    }
}

void opal_cuda_output(int output_id, const char *format, ...)
{
    if (output_id >= 0 && output_id <= OPAL_DATATYPE_CUDA_DEBUG_LEVEL) {
        va_list arglist;
        fprintf( stderr, "[Debug %d]: ", output_id );
        va_start(arglist, format);
        vfprintf(stderr, format, arglist);
        va_end(arglist);
    }
}

int32_t opal_ddt_cuda_kernel_init(void)
{
    uint32_t i, j;
    int device;
    hipError_t res;

    res = hipGetDevice(&device);
    if( hipSuccess != res ) {
        opal_cuda_output(0, "Cannot retrieve the device being used. Drop CUDA support!\n");
        return OPAL_ERROR;
    }    

    cuda_free_list = init_cuda_free_list();
    
    /* init device */
    cuda_devices = (ddt_cuda_device_t *)malloc(sizeof(ddt_cuda_device_t)*NB_GPUS);
    for (i = 0; i < NB_GPUS; i++) {
        unsigned char *gpu_ptr = NULL;
        if (hipMalloc((void **)(&gpu_ptr), sizeof(char)*DT_CUDA_BUFFER_SIZE) != hipSuccess) {
            DT_CUDA_DEBUG( opal_cuda_output( 0, "hipMalloc is failed in GPU %d\n", i); );
            return OPAL_ERROR;
        }
        DT_CUDA_DEBUG ( opal_cuda_output(2, "DDT engine hipMalloc buffer %p in GPU %d\n", gpu_ptr, i););
        hipMemset(gpu_ptr, 0, sizeof(char)*DT_CUDA_BUFFER_SIZE);
        cuda_devices[i].gpu_buffer = gpu_ptr;
        
        cuda_devices[i].buffer_free_size = DT_CUDA_BUFFER_SIZE;
        ddt_cuda_buffer_t *p = obj_ddt_cuda_buffer_new();
        p->size = DT_CUDA_BUFFER_SIZE;
        p->gpu_addr = gpu_ptr;
        cuda_devices[i].buffer_free.head = p;
        cuda_devices[i].buffer_free.tail = cuda_devices[i].buffer_free.head;
        cuda_devices[i].buffer_free.nb_elements = 1;
        
        cuda_devices[i].buffer_used.head = NULL;
        cuda_devices[i].buffer_used.tail = NULL;
        cuda_devices[i].buffer_used_size = 0;
        cuda_devices[i].buffer_used.nb_elements = 0;
    
        /* init cuda stream */
        ddt_cuda_stream_t *cuda_streams = (ddt_cuda_stream_t *)malloc(sizeof(ddt_cuda_stream_t));
        ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
        for (j = 0; j < NB_STREAMS; j++) {
            hipStreamCreate(&(cuda_streams->opal_cuda_stream[j]));
            cuda_iov_pipeline_block = (ddt_cuda_iov_pipeline_block_t *)malloc(sizeof(ddt_cuda_iov_pipeline_block_t));
            hipHostMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h)), sizeof(ddt_cuda_iov_dist_non_cached_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            hipMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d)), sizeof(ddt_cuda_iov_dist_non_cached_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            hipHostMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_dist_cached_h)), sizeof(ddt_cuda_iov_dist_cached_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            hipHostMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_contig_buf_h)), sizeof(uintptr_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            hipMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_contig_buf_d)), sizeof(uintptr_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            cuda_iov_pipeline_block->hip_stream = &(cuda_streams->opal_cuda_stream[0]);
            cuda_iov_pipeline_block->cuda_stream_id = 0;
            hipEventCreate(&(cuda_iov_pipeline_block->cuda_event), hipEventDisableTiming);
            cuda_devices[i].cuda_iov_pipeline_block[j] = cuda_iov_pipeline_block;
        }
        cuda_streams->current_stream_id = 0;
        cuda_devices[i].cuda_streams = cuda_streams;
        hipEventCreate(&(cuda_devices[i].memcpy_event), hipEventDisableTiming);
    }
    current_cuda_device = &(cuda_devices[0]);
    
    /* init cuda_iov */
    cuda_iov_count = CUDA_NB_IOV;
    
    // /* init size for double, float, char */
    // ALIGNMENT_DOUBLE = sizeof(double);
    // ALIGNMENT_FLOAT = sizeof(float);
    // ALIGNMENT_CHAR = sizeof(char);
    
    hipDeviceSynchronize();
    return OPAL_SUCCESS;
}

int32_t opal_ddt_cuda_kernel_fini(void)
{
    uint32_t i, j;
    
    for (i = 0; i < NB_GPUS; i++) {
        /* free gpu buffer */
        hipFree(cuda_devices[i].gpu_buffer);   
        /* destory cuda stream and iov*/
        ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
        for (j = 0; j < NB_STREAMS; j++) {
            hipStreamDestroy(cuda_devices[i].cuda_streams->opal_cuda_stream[j]);
            cuda_iov_pipeline_block = cuda_devices[i].cuda_iov_pipeline_block[j];
            if (cuda_iov_pipeline_block != NULL) {
                hipHostFree(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h);
                hipFree(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d);
                hipHostFree(cuda_iov_pipeline_block->cuda_iov_dist_cached_h);
                hipHostFree(cuda_iov_pipeline_block->cuda_iov_contig_buf_h);
                hipFree(cuda_iov_pipeline_block->cuda_iov_contig_buf_d);
                hipEventDestroy(cuda_iov_pipeline_block->cuda_event);
                cuda_iov_pipeline_block->hip_stream = NULL;
                cuda_iov_pipeline_block->cuda_stream_id = -1;
                free(cuda_iov_pipeline_block);
                cuda_iov_pipeline_block = NULL;
            }
        }
        free(cuda_devices[i].cuda_streams);
        cuda_devices[i].cuda_streams = NULL;
        hipEventDestroy(cuda_devices[i].memcpy_event);
    }
    current_cuda_device = NULL;
    return OPAL_SUCCESS;
}

void* opal_ddt_cached_cuda_iov_init(uint32_t size) 
{
#if OPAL_DATATYPE_CUDA_IOV_CACHE 
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)malloc(sizeof(ddt_cuda_iov_total_cached_t));
    ddt_cuda_iov_dist_cached_t *tmp_cuda_iov_d = NULL;
    hipMalloc((void **)(&tmp_cuda_iov_d), sizeof(ddt_cuda_iov_dist_cached_t) * size);
    uint32_t *tmp_nb_bytes = (uint32_t *)malloc(sizeof(uint32_t) * size);
    if (tmp != NULL && tmp_cuda_iov_d != NULL && tmp_nb_bytes != NULL) {
        tmp->cuda_iov_dist_d = tmp_cuda_iov_d;
        tmp->cuda_iov_count = size;
        tmp->cuda_iov_is_cached = 0;
        tmp->nb_bytes_h = tmp_nb_bytes;
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Malloc cuda_iov_dist_cached for ddt is successed, cached cuda iov %p, cuda_iov_d %p, nb_bytes_h %p, size %d.\n", tmp, tmp_cuda_iov_d, tmp_nb_bytes, size); );
        return tmp;
    } else {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "Malloc cuda_iov_dist_cached for ddt is failed.\n"); );
        return NULL;
    }
#else
    DT_CUDA_DEBUG( opal_cuda_output( 2, "cuda iov cache is not enabled.\n"); );
    return NULL;
#endif /* OPAL_DATATYPE_CUDA_IOV_CACHE */
}

void opal_ddt_cached_cuda_iov_fini(void* cached_cuda_iov) 
{
#if OPAL_DATATYPE_CUDA_IOV_CACHE
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *) cached_cuda_iov;
    if (tmp != NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Free cuda_iov_dist for ddt is successed %p.\n", tmp); );
        if (tmp->cuda_iov_dist_d != NULL) {
            hipFree(tmp->cuda_iov_dist_d);
            tmp->cuda_iov_dist_d = NULL;
        }
        if (tmp->nb_bytes_h != NULL) {
            free(tmp->nb_bytes_h);
            tmp->nb_bytes_h = NULL;
        }
        free(tmp);
        tmp = NULL;
    }
#endif /* OPAL_DATATYPE_CUDA_IOV_CACHE */
}

void opal_ddt_get_cached_cuda_iov(struct opal_convertor_t *convertor, ddt_cuda_iov_total_cached_t **cached_cuda_iov)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    if (datatype->cached_cuda_iov == NULL) {
        datatype->cached_cuda_iov = opal_ddt_cached_cuda_iov_init(NUM_CUDA_IOV_PER_DDT);
    }
    *cached_cuda_iov = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;                   
}

void opal_ddt_set_cuda_iov_cached(struct opal_convertor_t *convertor, uint32_t cuda_iov_count)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    assert(datatype->cached_cuda_iov != NULL);
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;
    tmp->cuda_iov_count = cuda_iov_count;
    tmp->cuda_iov_is_cached = 1;
}

uint8_t opal_ddt_cuda_iov_is_cached(struct opal_convertor_t *convertor)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    assert(datatype->cached_cuda_iov != NULL);
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;
    return tmp->cuda_iov_is_cached;
}

void opal_ddt_set_cuda_iov_position(struct opal_convertor_t *convertor, size_t ddt_offset, const uint32_t *cached_cuda_iov_nb_bytes_list_h, const uint32_t cuda_iov_count)
{
    int i;
    size_t iov_size = 0;
    convertor->current_iov_partial_length = 0;
    convertor->current_cuda_iov_pos = 0;
    if (ddt_offset == 0) {
       return;
    }
    for(i = 0; i < cuda_iov_count; i++) {
        iov_size += cached_cuda_iov_nb_bytes_list_h[i];
        if (iov_size > ddt_offset) {
            convertor->current_iov_partial_length = iov_size - ddt_offset;
            convertor->current_cuda_iov_pos = i;
            break;
        } else if (iov_size == ddt_offset){
            convertor->current_iov_partial_length = 0;
            convertor->current_cuda_iov_pos = i+1;
            break;
        }
    }
}

void opal_ddt_check_cuda_iov_is_full(struct opal_convertor_t *convertor, uint32_t cuda_iov_count)
{
#if 0
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    assert(datatype->cached_cuda_iov_dist != NULL);
    if (datatype->cached_cuda_iov_count < cuda_iov_count) {
        printf("cuda count %d, new count %d\n", datatype->cached_cuda_iov_count, cuda_iov_count);
  //      assert(0);
        void *old_iov = datatype->cached_cuda_iov_dist;
        void *new_iov = opal_ddt_cuda_iov_dist_init(datatype->cached_cuda_iov_count + NUM_CUDA_IOV_PER_DDT);
        assert(new_iov != NULL);
        hipMemcpy(new_iov, old_iov, datatype->cached_cuda_iov_count * sizeof(ddt_cuda_iov_dist_cached_t), hipMemcpyDeviceToDevice);
        datatype->cached_cuda_iov_dist = new_iov;
        datatype->cached_cuda_iov_count += NUM_CUDA_IOV_PER_DDT;
        opal_ddt_cuda_iov_dist_fini(old_iov);
    }
#endif
}

int32_t opal_ddt_cuda_is_gpu_buffer(const void *ptr)
{
    int res;
    hipMemoryType memType;
    hipDeviceptr_t dbuf = (hipDeviceptr_t)ptr;
    res = hipPointerGetAttribute(&memType, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dbuf);
    if (res != hipSuccess) {
        /* If we cannot determine it is device pointer,
         * just assume it is not. */
        printf("!!!!!!! %p is not a gpu buffer. Take no-CUDA path!\n", ptr);
        return 0;
    }
    /* Anything but hipMemoryTypeDevice is not a GPU memory */
    return (memType == hipMemoryTypeDevice) ? 1 : 0;
}

void* opal_ddt_cuda_malloc_gpu_buffer(size_t size, int gpu_id)
{
    int dev_id;
    hipGetDevice(&dev_id);
    ddt_cuda_device_t *device = &cuda_devices[gpu_id];
    if (device->buffer_free_size < size) {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "No GPU buffer at dev_id %d.\n", dev_id); );
        return NULL;
    }
    ddt_cuda_buffer_t *ptr = device->buffer_free.head;
    while (ptr != NULL) {
        if (ptr->size < size) {  /* Not enough room in this buffer, check next */
            ptr = ptr->next;
            continue;
        }
        void *addr = ptr->gpu_addr;
        ptr->size -= size;
        if (ptr->size == 0) {
            cuda_list_delete(&device->buffer_free, ptr);
            obj_ddt_cuda_buffer_reset(ptr);
            /* hold on this ptr object, we will reuse it right away */
        } else {
            ptr->gpu_addr += size;
            ptr = cuda_list_pop_tail(cuda_free_list);
            if( NULL == ptr )
                ptr = obj_ddt_cuda_buffer_new();
        }
        assert(NULL != ptr);
        ptr->size = size;
        ptr->gpu_addr = (unsigned char*)addr;
        cuda_list_push_head(&device->buffer_used, ptr);
        device->buffer_used_size += size;
        device->buffer_free_size -= size;
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Malloc GPU buffer %p, dev_id %d.\n", addr, dev_id); );
        return addr;
    }
    return NULL;
}

void opal_ddt_cuda_free_gpu_buffer(void *addr, int gpu_id)
{
    ddt_cuda_device_t *device = &cuda_devices[gpu_id];
    ddt_cuda_buffer_t *ptr = device->buffer_used.head;

    /* Find the holder of this GPU allocation */
    for( ; (NULL != ptr) && (ptr->gpu_addr != addr); ptr = ptr->next );
    if (NULL == ptr) {  /* we could not find it. something went wrong */
        DT_CUDA_DEBUG( opal_cuda_output( 0, "addr %p is not managed.\n", addr); );
        return;
    }
    cuda_list_delete(&device->buffer_used, ptr);
    /* Insert the element in the list of free buffers ordered by the addr */
    ddt_cuda_buffer_t *ptr_next = device->buffer_free.head;
    while (ptr_next != NULL) {
        if (ptr_next->gpu_addr > addr) {
            break;
        }
        ptr_next = ptr_next->next;
    }
    if (ptr_next == NULL) {  /* buffer_free is empty, or insert to last one */
        cuda_list_push_tail(&device->buffer_free, ptr);
    } else {
        cuda_list_insert_before(&device->buffer_free, ptr, ptr_next);
    }
    size_t size = ptr->size;
    cuda_list_item_merge_by_addr(&device->buffer_free, ptr);
    device->buffer_free_size += size;
    device->buffer_used_size -= size;
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Free GPU buffer %p.\n", addr); );
}

void opal_cuda_check_error(hipError_t err)
{
    if (err != hipSuccess) {
        DT_CUDA_DEBUG( opal_cuda_output(0, "CUDA calls error %s\n", hipGetErrorString(err)); );
    }
}

void opal_ddt_cuda_d2dcpy_async(void* dst, const void* src, size_t count)
{
    hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, current_cuda_device->cuda_streams->opal_cuda_stream[0]);
}

void opal_ddt_cuda_d2dcpy(void* dst, const void* src, size_t count)
{
    hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, current_cuda_device->cuda_streams->opal_cuda_stream[0]);
    hipStreamSynchronize(current_cuda_device->cuda_streams->opal_cuda_stream[0]);
}

void opal_dump_cuda_list(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *ptr = NULL;
    ptr = list->head;
    DT_CUDA_DEBUG( opal_cuda_output( 2, "DUMP cuda list %p, nb_elements %d\n", list, list->nb_elements); );
    while (ptr != NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 2, "\titem addr %p, size %ld.\n", ptr->gpu_addr, ptr->size); );
        ptr = ptr->next;
    }
}
