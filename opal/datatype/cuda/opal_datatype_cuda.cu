#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdarg.h> 


ddt_cuda_list_t *cuda_free_list;
ddt_cuda_device_t *cuda_devices;
ddt_cuda_device_t *current_cuda_device;
struct iovec cuda_iov[CUDA_NB_IOV];
uint32_t cuda_iov_count;

//uint8_t ALIGNMENT_DOUBLE, ALIGNMENT_FLOAT, ALIGNMENT_CHAR;


static inline ddt_cuda_buffer_t* obj_ddt_cuda_buffer_new()
{
    ddt_cuda_buffer_t *p = (ddt_cuda_buffer_t *)malloc(sizeof(ddt_cuda_buffer_t));
    p->next = NULL;
    p->prev = NULL;
    p->size = 0;
    p->gpu_addr = NULL;
    return p; 
}

static inline void obj_ddt_cuda_buffer_chop(ddt_cuda_buffer_t *p)
{
    p->next = NULL;
    p->prev = NULL;
}

static inline void obj_ddt_cuda_buffer_reset(ddt_cuda_buffer_t *p)
{
    p->size = 0;
    p->gpu_addr = NULL;
}

static ddt_cuda_list_t* init_cuda_free_list()
{
    ddt_cuda_list_t *list = NULL;
    ddt_cuda_buffer_t *p, *prev;
    int i;
    list = (ddt_cuda_list_t *)malloc(sizeof(ddt_cuda_list_t));
    p = obj_ddt_cuda_buffer_new();
    list->head = p;
    prev = p;
    for (i = 1; i < DT_CUDA_FREE_LIST_SIZE; i++) {
        p = obj_ddt_cuda_buffer_new();
        prev->next = p;
        p->prev = prev;
        prev = p;
    }
    list->tail = p;
    list->nb_elements = DT_CUDA_FREE_LIST_SIZE;
    return list;
} 

static inline ddt_cuda_buffer_t* cuda_list_pop_tail(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *p = NULL;
    p = list->tail;
    if (p == NULL) {
        return p;
    } else {
        list->nb_elements --;
        if (list->head == p) {
            list->head = NULL;
            list->tail = NULL;
        } else {
            list->tail = p->prev;
            p->prev->next = NULL;
            obj_ddt_cuda_buffer_chop(p);
        }
        return p;
    }
}

static inline void cuda_list_push_head(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_head = list->head;
    assert(item->next == NULL && item->prev == NULL);
    list->head = item;
    item->next = orig_head;
    if (orig_head == NULL) {
        list->tail = item;
    } else {
        orig_head->prev = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_push_tail(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    ddt_cuda_buffer_t * orig_tail = list->tail;
    assert(item->next == NULL && item->prev == NULL);
    list->tail = item;
    item->prev = orig_tail;
    if (orig_tail == NULL) {
        list->head = item;
    } else {
        orig_tail->next = item;
    }
    list->nb_elements ++;
}

static inline void cuda_list_delete(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item)
{
    if (item->prev == NULL && item->next == NULL) {
        list->head = NULL;
        list->tail = NULL;
    }else if (item->prev == NULL && item->next != NULL) {
        list->head = item->next;
        item->next->prev = NULL;
    } else if (item->next == NULL && item->prev != NULL) {
        list->tail = item->prev;
        item->prev->next = NULL;
    } else {
        item->prev->next = item->next;
        item->next->prev = item->prev;
    }
    list->nb_elements --;
    obj_ddt_cuda_buffer_chop(item);
}

static inline void cuda_list_insert_before(ddt_cuda_list_t *list, ddt_cuda_buffer_t *item, ddt_cuda_buffer_t *next)
{
    assert(item->next == NULL && item->prev == NULL);
    item->next = next;
    item->prev = next->prev;
    next->prev = item;
    if (list->head == next) {
        list->head = item;
    }
    list->nb_elements ++;
}

/**
 * Collapse the list of free buffers by mergining consecutive buffers. As the property of this list
 * is continously maintained, we only have to parse it up to the newest inserted elements.
 */
static inline void cuda_list_item_merge_by_addr(ddt_cuda_list_t *list, ddt_cuda_buffer_t* last)
{
    ddt_cuda_buffer_t *current = list->head;
    ddt_cuda_buffer_t *next = NULL;
    void* stop_addr = last->gpu_addr;

    while(1) {  /* loop forever, the exit conditions are inside */
        if( NULL == (next = current->next) ) return;
        if ((current->gpu_addr + current->size) == next->gpu_addr) {
            current->size += next->size;
            cuda_list_delete(list, next);
            free(next);  /* release the element, and try to continue merging */
            continue;
        }
        current = current->next;
        if( NULL == current ) return;
        if( current->gpu_addr > stop_addr ) return;
    }
}

void opal_cuda_output(int output_id, const char *format, ...)
{
    if (output_id >= 0 && output_id <= OPAL_DATATYPE_CUDA_DEBUG_LEVEL) {
        va_list arglist;
        fprintf( stderr, "[Debug %d]: ", output_id );
        va_start(arglist, format);
        vfprintf(stderr, format, arglist);
        va_end(arglist);
    }
}

int32_t opal_ddt_cuda_kernel_init(void)
{
    uint32_t i, j;
    int device;
    hipError_t res;

    res = hipGetDevice(&device);
    if( hipSuccess != res ) {
        opal_cuda_output(0, "Cannot retrieve the device being used. Drop CUDA support!\n");
        return OPAL_ERROR;
    }    

    cuda_free_list = init_cuda_free_list();
    
    /* init device */
    cuda_devices = (ddt_cuda_device_t *)malloc(sizeof(ddt_cuda_device_t)*NB_GPUS);
    for (i = 0; i < NB_GPUS; i++) {
        unsigned char *gpu_ptr = NULL;
        if (hipMalloc((void **)(&gpu_ptr), sizeof(char)*DT_CUDA_BUFFER_SIZE) != hipSuccess) {
            DT_CUDA_DEBUG( opal_cuda_output( 0, "hipMalloc is failed in GPU %d\n", i); );
            return OPAL_ERROR;
        }
        DT_CUDA_DEBUG ( opal_cuda_output(2, "DDT engine hipMalloc buffer %p in GPU %d\n", gpu_ptr, i););
        hipMemset(gpu_ptr, 0, sizeof(char)*DT_CUDA_BUFFER_SIZE);
        cuda_devices[i].gpu_buffer = gpu_ptr;
        
        cuda_devices[i].buffer_free_size = DT_CUDA_BUFFER_SIZE;
        ddt_cuda_buffer_t *p = obj_ddt_cuda_buffer_new();
        p->size = DT_CUDA_BUFFER_SIZE;
        p->gpu_addr = gpu_ptr;
        cuda_devices[i].buffer_free.head = p;
        cuda_devices[i].buffer_free.tail = cuda_devices[i].buffer_free.head;
        cuda_devices[i].buffer_free.nb_elements = 1;
        
        cuda_devices[i].buffer_used.head = NULL;
        cuda_devices[i].buffer_used.tail = NULL;
        cuda_devices[i].buffer_used_size = 0;
        cuda_devices[i].buffer_used.nb_elements = 0;
    
        /* init cuda stream */
        ddt_cuda_stream_t *cuda_streams = (ddt_cuda_stream_t *)malloc(sizeof(ddt_cuda_stream_t));
        ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
        for (j = 0; j < NB_STREAMS; j++) {
            hipStreamCreate(&(cuda_streams->opal_cuda_stream[j]));
            cuda_iov_pipeline_block = (ddt_cuda_iov_pipeline_block_t *)malloc(sizeof(ddt_cuda_iov_pipeline_block_t));
            hipHostMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h)), sizeof(ddt_cuda_iov_dist_cached_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            hipMalloc((void **)(&(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d)), sizeof(ddt_cuda_iov_dist_cached_t) * CUDA_MAX_NB_BLOCKS * CUDA_IOV_MAX_TASK_PER_BLOCK);
            if (j == 0) {
                cuda_iov_pipeline_block->cuda_iov_dist_cached_h = (ddt_cuda_iov_dist_cached_t *)malloc(sizeof(ddt_cuda_iov_dist_cached_t) * NUM_CUDA_IOV_PER_DDT);
            } else {
                cuda_iov_pipeline_block->cuda_iov_dist_cached_h = NULL;
            }
            cuda_iov_pipeline_block->hip_stream = &(cuda_streams->opal_cuda_stream[0]);
            cuda_iov_pipeline_block->cuda_stream_id = 0;
            hipEventCreate(&(cuda_iov_pipeline_block->cuda_event), hipEventDisableTiming);
            cuda_devices[i].cuda_iov_pipeline_block[j] = cuda_iov_pipeline_block;
        }
        cuda_streams->current_stream_id = 0;
        cuda_devices[i].cuda_streams = cuda_streams;
        hipEventCreate(&(cuda_devices[i].memcpy_event), hipEventDisableTiming);
    }
    current_cuda_device = &(cuda_devices[0]);
    
    /* init cuda_iov */
    cuda_iov_count = CUDA_NB_IOV;
    
    // /* init size for double, float, char */
    // ALIGNMENT_DOUBLE = sizeof(double);
    // ALIGNMENT_FLOAT = sizeof(float);
    // ALIGNMENT_CHAR = sizeof(char);
    
    hipDeviceSynchronize();
    return OPAL_SUCCESS;
}

int32_t opal_ddt_cuda_kernel_fini(void)
{
    uint32_t i, j;
    
    for (i = 0; i < NB_GPUS; i++) {
        /* free gpu buffer */
        hipFree(cuda_devices[i].gpu_buffer);   
        /* destory cuda stream and iov*/
        ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
        for (j = 0; j < NB_STREAMS; j++) {
            hipStreamDestroy(cuda_devices[i].cuda_streams->opal_cuda_stream[j]);
            cuda_iov_pipeline_block = cuda_devices[i].cuda_iov_pipeline_block[j];
            if (cuda_iov_pipeline_block != NULL) {
                if (cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h != NULL) {
                    hipHostFree(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h);
                    cuda_iov_pipeline_block->cuda_iov_dist_non_cached_h = NULL;
                }
                if (cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d != NULL) {
                    hipFree(cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d);
                    cuda_iov_pipeline_block->cuda_iov_dist_non_cached_d = NULL;
                }
                if (cuda_iov_pipeline_block->cuda_iov_dist_cached_h != NULL) {
                    free(cuda_iov_pipeline_block->cuda_iov_dist_cached_h);
                    cuda_iov_pipeline_block->cuda_iov_dist_cached_h = NULL;
                }
                hipEventDestroy(cuda_iov_pipeline_block->cuda_event);
                cuda_iov_pipeline_block->hip_stream = NULL;
                cuda_iov_pipeline_block->cuda_stream_id = -1;
                free(cuda_iov_pipeline_block);
                cuda_iov_pipeline_block = NULL;
            }
        }
        free(cuda_devices[i].cuda_streams);
        cuda_devices[i].cuda_streams = NULL;
        hipEventDestroy(cuda_devices[i].memcpy_event);
    }
    current_cuda_device = NULL;
    return OPAL_SUCCESS;
}

void* opal_ddt_cached_cuda_iov_init(uint32_t size) 
{
#if OPAL_DATATYPE_CUDA_IOV_CACHE 
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)malloc(sizeof(ddt_cuda_iov_total_cached_t));
    uint32_t *tmp_nb_bytes = (uint32_t *)malloc(sizeof(uint32_t) * size);
    if (tmp != NULL && tmp_nb_bytes != NULL) {
        tmp->cuda_iov_dist_d = NULL;
        tmp->cuda_iov_count = size;
        tmp->cuda_iov_is_cached = 0;
        tmp->nb_bytes_h = tmp_nb_bytes;
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Malloc cuda_iov_dist_cached for ddt is successed, cached cuda iov %p, nb_bytes_h %p, size %d.\n", tmp, tmp_nb_bytes, size); );
        return tmp;
    } else {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "Malloc cuda_iov_dist_cached for ddt is failed.\n"); );
        return NULL;
    }
#else
    DT_CUDA_DEBUG( opal_cuda_output( 2, "cuda iov cache is not enabled.\n"); );
    return NULL;
#endif /* OPAL_DATATYPE_CUDA_IOV_CACHE */
}

void opal_ddt_cached_cuda_iov_fini(void* cached_cuda_iov) 
{
#if OPAL_DATATYPE_CUDA_IOV_CACHE
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *) cached_cuda_iov;
    if (tmp != NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Free cuda_iov_dist for ddt is successed %p.\n", tmp); );
        if (tmp->cuda_iov_dist_d != NULL) {
            hipFree(tmp->cuda_iov_dist_d);
            tmp->cuda_iov_dist_d = NULL;
        }
        if (tmp->nb_bytes_h != NULL) {
            free(tmp->nb_bytes_h);
            tmp->nb_bytes_h = NULL;
        }
        free(tmp);
        tmp = NULL;
    }
#endif /* OPAL_DATATYPE_CUDA_IOV_CACHE */
}

static inline int32_t opal_ddt_cached_cuda_iov_isfull(ddt_cuda_iov_total_cached_t *cached_cuda_iov, ddt_cuda_iov_dist_cached_t **cuda_iov_dist_h, uint32_t nb_blocks_used)
{
    if (nb_blocks_used < cached_cuda_iov->cuda_iov_count) {
        return 0;
    } else {
realloc_cuda_iov:
        cached_cuda_iov->nb_bytes_h = (uint32_t *)realloc(cached_cuda_iov->nb_bytes_h, sizeof(uint32_t)*cached_cuda_iov->cuda_iov_count*2);
        assert(cached_cuda_iov->nb_bytes_h != NULL);
        cached_cuda_iov->cuda_iov_count *= 2;
        if (nb_blocks_used >= cached_cuda_iov->cuda_iov_count) {
            goto realloc_cuda_iov;
        }
        return 1;
    }
}

/* cached_cuda_iov_d is not ready until explicitlt sync with cuda stream 0 
*/
int32_t opal_ddt_cache_cuda_iov(opal_convertor_t* pConvertor, uint32_t *cuda_iov_count)
{
    uint32_t i, j;
    uint32_t count_desc, nb_blocks_per_description, residue_desc;
    uint32_t thread_per_block, nb_blocks_used;
    size_t length_per_iovec;
    uint8_t alignment;
    ddt_cuda_iov_pipeline_block_t *cuda_iov_pipeline_block = NULL;
    ddt_cuda_iov_total_cached_t* cached_cuda_iov = NULL;
    ddt_cuda_iov_dist_cached_t *cached_cuda_iov_dist_d = NULL;
    ddt_cuda_iov_dist_cached_t *cuda_iov_dist_h = NULL;
    hipStream_t *cuda_stream_iov = NULL;
    const struct iovec *ddt_iov = NULL;
    uint32_t ddt_iov_count = 0;
    size_t ncontig_disp_base;
    size_t contig_disp = 0;
    uint32_t *cached_cuda_iov_nb_bytes_list_h = NULL;
    
    opal_datatype_t *datatype = (opal_datatype_t *)pConvertor->pDesc;
    
    opal_convertor_raw_cached( pConvertor, &ddt_iov, &ddt_iov_count);
    if (ddt_iov == NULL) {
        DT_CUDA_DEBUG ( opal_cuda_output(0, "Can not get ddt iov\n"););
        return OPAL_ERROR;
    }
    
    
    cached_cuda_iov = (ddt_cuda_iov_total_cached_t *)opal_ddt_cached_cuda_iov_init(NUM_CUDA_IOV_PER_DDT);
    if (cached_cuda_iov == NULL) {
        DT_CUDA_DEBUG ( opal_cuda_output(0, "Can not init cuda iov\n"););
        return OPAL_ERROR;
    }
    cached_cuda_iov_nb_bytes_list_h = cached_cuda_iov->nb_bytes_h;
    nb_blocks_used = 0;
    cuda_iov_pipeline_block = current_cuda_device->cuda_iov_pipeline_block[0];
    cuda_iov_dist_h = cuda_iov_pipeline_block->cuda_iov_dist_cached_h;
    cuda_stream_iov = cuda_iov_pipeline_block->hip_stream;
    thread_per_block = CUDA_WARP_SIZE * 5;

    for (i = 0; i < ddt_iov_count; i++) {
        length_per_iovec = ddt_iov[i].iov_len;
        ncontig_disp_base = (size_t)(ddt_iov[i].iov_base);
    
        /* block size is either multiple of ALIGNMENT_DOUBLE or residule */
        alignment = ALIGNMENT_DOUBLE;

        count_desc = length_per_iovec / alignment;
        residue_desc = length_per_iovec % alignment;
        nb_blocks_per_description = (count_desc + thread_per_block - 1) / thread_per_block;
        DT_CUDA_DEBUG ( opal_cuda_output(10, "Cache cuda IOV description %d, size %d, residue %d, alignment %d, nb_block_aquired %d\n", i, count_desc, residue_desc, alignment, nb_blocks_per_description); );
        if (opal_ddt_cached_cuda_iov_isfull(cached_cuda_iov, &(cuda_iov_pipeline_block->cuda_iov_dist_cached_h), nb_blocks_used + nb_blocks_per_description + 1)) {
            cached_cuda_iov_nb_bytes_list_h = cached_cuda_iov->nb_bytes_h;
            cuda_iov_dist_h = (ddt_cuda_iov_dist_cached_t *)realloc(cuda_iov_dist_h, sizeof(ddt_cuda_iov_dist_cached_t)*cached_cuda_iov->cuda_iov_count);
            assert(cuda_iov_dist_h != NULL);
            cuda_iov_pipeline_block->cuda_iov_dist_cached_h = cuda_iov_dist_h;
        }
        
        for (j = 0; j < nb_blocks_per_description; j++) {
            cuda_iov_dist_h[nb_blocks_used].ncontig_disp = ncontig_disp_base + j * thread_per_block * alignment;
            cuda_iov_dist_h[nb_blocks_used].contig_disp = contig_disp;
            if ( (j+1) * thread_per_block <= count_desc) {
                cached_cuda_iov_nb_bytes_list_h[nb_blocks_used] = thread_per_block * alignment;
            } else {
                cached_cuda_iov_nb_bytes_list_h[nb_blocks_used] = (count_desc - j*thread_per_block) * alignment; 
            }
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
            assert(cached_cuda_iov_nb_bytes_list_h[nb_blocks_used] > 0);
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
            contig_disp += cached_cuda_iov_nb_bytes_list_h[nb_blocks_used];
            DT_CUDA_DEBUG( opal_cuda_output(12, "Cache cuda IOV \tblock %d, ncontig_disp %ld, contig_disp %ld, nb_bytes %ld\n", nb_blocks_used, cuda_iov_dist_h[nb_blocks_used].ncontig_disp, cuda_iov_dist_h[nb_blocks_used].contig_disp, cached_cuda_iov_nb_bytes_list_h[nb_blocks_used]); );
            nb_blocks_used ++;
         //   assert (nb_blocks_used < NUM_CUDA_IOV_PER_DDT);
        }
    
        /* handle residue */
        if (residue_desc != 0) {
            cuda_iov_dist_h[nb_blocks_used].ncontig_disp = ncontig_disp_base + length_per_iovec / alignment * alignment;
            cuda_iov_dist_h[nb_blocks_used].contig_disp = contig_disp;
            cached_cuda_iov_nb_bytes_list_h[nb_blocks_used] = length_per_iovec - length_per_iovec / alignment * alignment;
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
            assert(cached_cuda_iov_nb_bytes_list_h[nb_blocks_used] > 0);
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
            contig_disp += cached_cuda_iov_nb_bytes_list_h[nb_blocks_used];
            DT_CUDA_DEBUG( opal_cuda_output(12, "Cache cuda IOV \tblock %d, ncontig_disp %ld, contig_disp %ld, nb_bytes %ld\n", nb_blocks_used, cuda_iov_dist_h[nb_blocks_used].ncontig_disp, cuda_iov_dist_h[nb_blocks_used].contig_disp, cached_cuda_iov_nb_bytes_list_h[nb_blocks_used]); );
            nb_blocks_used ++;
#if defined (OPAL_DATATYPE_CUDA_DEBUG)
            //assert (nb_blocks_used < NUM_CUDA_IOV_PER_DDT);
#endif /* OPAL_DATATYPE_CUDA_DEBUG */
        }
    }
    /* use additional entry to store the size of entire contiguous buffer needed for one ddt */
    cuda_iov_dist_h[nb_blocks_used].contig_disp = contig_disp;
    hipMalloc((void **)(&cached_cuda_iov_dist_d), sizeof(ddt_cuda_iov_dist_cached_t) * (nb_blocks_used+1));
    if (cached_cuda_iov_dist_d == NULL) {
        DT_CUDA_DEBUG ( opal_cuda_output(0, "Can not malloc cuda iov in GPU\n"););
        return OPAL_ERROR;
    }
    hipMemcpyAsync(cached_cuda_iov_dist_d, cuda_iov_dist_h, sizeof(ddt_cuda_iov_dist_cached_t)*(nb_blocks_used+1), hipMemcpyHostToDevice, *cuda_stream_iov);
    cached_cuda_iov->cuda_iov_dist_d = cached_cuda_iov_dist_d;
    datatype->cached_cuda_iov = (unsigned char*)cached_cuda_iov;
    *cuda_iov_count = nb_blocks_used;
    return OPAL_SUCCESS;
}

void opal_ddt_get_cached_cuda_iov(struct opal_convertor_t *convertor, ddt_cuda_iov_total_cached_t **cached_cuda_iov)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    if (datatype->cached_cuda_iov == NULL) {
        *cached_cuda_iov = NULL;
    } else {
        *cached_cuda_iov = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;
    }                 
}

void opal_ddt_set_cuda_iov_cached(struct opal_convertor_t *convertor, uint32_t cuda_iov_count)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    assert(datatype->cached_cuda_iov != NULL);
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;
    tmp->cuda_iov_count = cuda_iov_count;
    tmp->cuda_iov_is_cached = 1;
}

uint8_t opal_ddt_cuda_iov_is_cached(struct opal_convertor_t *convertor)
{
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    if (datatype->cached_cuda_iov == NULL) {
        return 0;
    }
    ddt_cuda_iov_total_cached_t *tmp = (ddt_cuda_iov_total_cached_t *)datatype->cached_cuda_iov;
    return tmp->cuda_iov_is_cached;
}

void opal_ddt_set_cuda_iov_position(struct opal_convertor_t *convertor, size_t ddt_offset, const uint32_t *cached_cuda_iov_nb_bytes_list_h, const uint32_t cuda_iov_count)
{
    int i;
    size_t iov_size = 0;
    size_t ddt_size;
    convertor->current_iov_partial_length = 0;
    convertor->current_cuda_iov_pos = 0;
    convertor->current_count = 0;
    if (ddt_offset == 0) {
       return;
    }
    opal_datatype_type_size(convertor->pDesc, &ddt_size);
    convertor->current_count = ddt_offset / ddt_size;
    ddt_offset = ddt_offset % ddt_size;
    for(i = 0; i < cuda_iov_count; i++) {
        iov_size += cached_cuda_iov_nb_bytes_list_h[i];
        if (iov_size > ddt_offset) {
            convertor->current_iov_partial_length = iov_size - ddt_offset;
            convertor->current_cuda_iov_pos = i;
            break;
        } else if (iov_size == ddt_offset){
            convertor->current_iov_partial_length = 0;
            convertor->current_cuda_iov_pos = i+1;
            break;
        }
    }
}

void opal_ddt_set_ddt_iov_position(struct opal_convertor_t *convertor, size_t ddt_offset, const struct iovec *ddt_iov, const uint32_t ddt_iov_count)
{
    int i;
    size_t iov_size = 0;
    size_t ddt_size;
    convertor->current_iov_partial_length = 0;
    convertor->current_iov_pos = 0;
    convertor->current_count = 0;
    if (ddt_offset == 0) {
       return;
    }
    opal_datatype_type_size(convertor->pDesc, &ddt_size);
    convertor->current_count = ddt_offset / ddt_size;
    ddt_offset = ddt_offset % ddt_size;
    for(i = 0; i < ddt_iov_count; i++) {
        iov_size += ddt_iov[i].iov_len;
        if (iov_size > ddt_offset) {
            convertor->current_iov_partial_length = iov_size - ddt_offset;
            convertor->current_iov_pos = i;
            break;
        } else if (iov_size == ddt_offset){
            convertor->current_iov_partial_length = 0;
            convertor->current_iov_pos = i+1;
            break;
        }
    }
}

void opal_ddt_check_cuda_iov_is_full(struct opal_convertor_t *convertor, uint32_t cuda_iov_count)
{
#if 0
    opal_datatype_t *datatype = (opal_datatype_t *)convertor->pDesc;
    assert(datatype->cached_cuda_iov_dist != NULL);
    if (datatype->cached_cuda_iov_count < cuda_iov_count) {
        printf("cuda count %d, new count %d\n", datatype->cached_cuda_iov_count, cuda_iov_count);
  //      assert(0);
        void *old_iov = datatype->cached_cuda_iov_dist;
        void *new_iov = opal_ddt_cuda_iov_dist_init(datatype->cached_cuda_iov_count + NUM_CUDA_IOV_PER_DDT);
        assert(new_iov != NULL);
        hipMemcpy(new_iov, old_iov, datatype->cached_cuda_iov_count * sizeof(ddt_cuda_iov_dist_cached_t), hipMemcpyDeviceToDevice);
        datatype->cached_cuda_iov_dist = new_iov;
        datatype->cached_cuda_iov_count += NUM_CUDA_IOV_PER_DDT;
        opal_ddt_cuda_iov_dist_fini(old_iov);
    }
#endif
}

int32_t opal_ddt_cuda_is_gpu_buffer(const void *ptr)
{
    int res;
    hipMemoryType memType;
    hipDeviceptr_t dbuf = (hipDeviceptr_t)ptr;
    res = hipPointerGetAttribute(&memType, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dbuf);
    if (res != hipSuccess) {
        /* If we cannot determine it is device pointer,
         * just assume it is not. */
        printf("!!!!!!! %p is not a gpu buffer. Take no-CUDA path!\n", ptr);
        return 0;
    }
    /* Anything but hipMemoryTypeDevice is not a GPU memory */
    return (memType == hipMemoryTypeDevice) ? 1 : 0;
}

void* opal_ddt_cuda_malloc_gpu_buffer(size_t size, int gpu_id)
{
    int dev_id;
    hipGetDevice(&dev_id);
    ddt_cuda_device_t *device = &cuda_devices[gpu_id];
    if (device->buffer_free_size < size) {
        DT_CUDA_DEBUG( opal_cuda_output( 0, "No GPU buffer at dev_id %d.\n", dev_id); );
        return NULL;
    }
    ddt_cuda_buffer_t *ptr = device->buffer_free.head;
    while (ptr != NULL) {
        if (ptr->size < size) {  /* Not enough room in this buffer, check next */
            ptr = ptr->next;
            continue;
        }
        void *addr = ptr->gpu_addr;
        ptr->size -= size;
        if (ptr->size == 0) {
            cuda_list_delete(&device->buffer_free, ptr);
            obj_ddt_cuda_buffer_reset(ptr);
            /* hold on this ptr object, we will reuse it right away */
        } else {
            ptr->gpu_addr += size;
            ptr = cuda_list_pop_tail(cuda_free_list);
            if( NULL == ptr )
                ptr = obj_ddt_cuda_buffer_new();
        }
        assert(NULL != ptr);
        ptr->size = size;
        ptr->gpu_addr = (unsigned char*)addr;
        cuda_list_push_head(&device->buffer_used, ptr);
        device->buffer_used_size += size;
        device->buffer_free_size -= size;
        DT_CUDA_DEBUG( opal_cuda_output( 2, "Malloc GPU buffer %p, dev_id %d.\n", addr, dev_id); );
        return addr;
    }
    return NULL;
}

void opal_ddt_cuda_free_gpu_buffer(void *addr, int gpu_id)
{
    ddt_cuda_device_t *device = &cuda_devices[gpu_id];
    ddt_cuda_buffer_t *ptr = device->buffer_used.head;

    /* Find the holder of this GPU allocation */
    for( ; (NULL != ptr) && (ptr->gpu_addr != addr); ptr = ptr->next );
    if (NULL == ptr) {  /* we could not find it. something went wrong */
        DT_CUDA_DEBUG( opal_cuda_output( 0, "addr %p is not managed.\n", addr); );
        return;
    }
    cuda_list_delete(&device->buffer_used, ptr);
    /* Insert the element in the list of free buffers ordered by the addr */
    ddt_cuda_buffer_t *ptr_next = device->buffer_free.head;
    while (ptr_next != NULL) {
        if (ptr_next->gpu_addr > addr) {
            break;
        }
        ptr_next = ptr_next->next;
    }
    if (ptr_next == NULL) {  /* buffer_free is empty, or insert to last one */
        cuda_list_push_tail(&device->buffer_free, ptr);
    } else {
        cuda_list_insert_before(&device->buffer_free, ptr, ptr_next);
    }
    size_t size = ptr->size;
    cuda_list_item_merge_by_addr(&device->buffer_free, ptr);
    device->buffer_free_size += size;
    device->buffer_used_size -= size;
    DT_CUDA_DEBUG( opal_cuda_output( 2, "Free GPU buffer %p.\n", addr); );
}

void opal_cuda_check_error(hipError_t err)
{
    if (err != hipSuccess) {
        DT_CUDA_DEBUG( opal_cuda_output(0, "CUDA calls error %s\n", hipGetErrorString(err)); );
    }
}

void opal_ddt_cuda_d2dcpy_async(void* dst, const void* src, size_t count)
{
    hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, current_cuda_device->cuda_streams->opal_cuda_stream[0]);
}

void opal_ddt_cuda_d2dcpy(void* dst, const void* src, size_t count)
{
    hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, current_cuda_device->cuda_streams->opal_cuda_stream[0]);
    hipStreamSynchronize(current_cuda_device->cuda_streams->opal_cuda_stream[0]);
}

void opal_dump_cuda_list(ddt_cuda_list_t *list)
{
    ddt_cuda_buffer_t *ptr = NULL;
    ptr = list->head;
    DT_CUDA_DEBUG( opal_cuda_output( 2, "DUMP cuda list %p, nb_elements %d\n", list, list->nb_elements); );
    while (ptr != NULL) {
        DT_CUDA_DEBUG( opal_cuda_output( 2, "\titem addr %p, size %ld.\n", ptr->gpu_addr, ptr->size); );
        ptr = ptr->next;
    }
}
