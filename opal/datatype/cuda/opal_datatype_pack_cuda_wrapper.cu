#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"

#include <stdio.h>

int32_t opal_generic_simple_pack_function_cuda( opal_convertor_t* pConvertor,
                                                struct iovec* iov, 
                                                uint32_t* out_size,
                                                size_t* max_data )
{
    uint32_t i;
    dt_elem_desc_t* description;
    const opal_datatype_t *pData = pConvertor->pDesc;
    uint32_t tasks_per_block, num_blocks;
    dt_stack_t* pStack;
    
    description = pConvertor->use_desc->desc;
    
    cuda_desc_h->stack_pos = pConvertor->stack_pos;
#if defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    cuda_desc_h->pBaseBuf = pConvertor->pBaseBuf;
#else
    cuda_desc_h->pBaseBuf = pBaseBuf_GPU;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
    cuda_desc_h->lb = pData->lb;
    cuda_desc_h->ub = pData->ub;
    cuda_desc_h->out_size = *out_size;
    cuda_desc_h->max_data = *max_data;
    cuda_desc_h->bConverted = pConvertor->bConverted;
    cuda_desc_h->local_size = pConvertor->local_size;
    cuda_desc_h->stack_size = pConvertor->stack_size;
    
    for (i = 0; i < pConvertor->stack_size; i++) {
        cuda_desc_h->pStack[i] = pConvertor->pStack[i];
    }
    if (cuda_desc_h->description_max_count != 0) {
        if (cuda_desc_h->description_max_count >= (pConvertor->use_desc->used+1)) {
            cuda_desc_h->description_count = pConvertor->use_desc->used+1;
        } else {
            hipFree(cuda_desc_h->description);
            cuda_desc_h->description = NULL;
            hipMalloc((void **)&(cuda_desc_h->description), sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1));
            cuda_desc_h->description_max_count = pConvertor->use_desc->used+1;
            cuda_desc_h->description_count = pConvertor->use_desc->used+1;
        }
        
    } else {
        hipMalloc((void **)&(cuda_desc_h->description), sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1));
        cuda_desc_h->description_max_count = pConvertor->use_desc->used+1;
        cuda_desc_h->description_count = pConvertor->use_desc->used+1;
    }
    hipMemcpy(cuda_desc_h->description, description, sizeof(dt_elem_desc_t)*(pConvertor->use_desc->used+1), hipMemcpyHostToDevice);
    
    // for (i = 0; i < pConvertor->use_desc->used+1; i++) {
    //     cuda_desc_h->description[i] = description[i];
    // }
    
    DBGPRINT("stack_size %d\n", pConvertor->stack_size);

    DBGPRINT("flags %d, types %d, count %d\n", description->elem.common.flags, description->elem.common.type, description->elem.count);
    
    for (i = 0; i < *out_size; i++) {
#if defined (OPAL_DATATYPE_CUDA_DRY_RUN)
        cuda_desc_h->iov[i].iov_base = iov[i].iov_base;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
        cuda_desc_h->iov[i].iov_len = iov[i].iov_len;
    }
    
    hipMemcpy(cuda_desc_d, cuda_desc_h, sizeof(ddt_cuda_desc_t), hipMemcpyHostToDevice);
    
    pStack = pConvertor->pStack + pConvertor->stack_pos;
    tasks_per_block = THREAD_PER_BLOCK * TASK_PER_THREAD;
    num_blocks = ((uint32_t)pStack->count + tasks_per_block - 1) / tasks_per_block;
    printf("launch kernel, count %d, num_blocks %d, total threads %d\n", (uint32_t)pStack->count, num_blocks, num_blocks*2*THREAD_PER_BLOCK);
    opal_generic_simple_pack_cuda_kernel<<<192,4*THREAD_PER_BLOCK>>>(cuda_desc_d);
#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    size_t position = pConvertor->pDesc->size;
    opal_convertor_set_position_nocheck(pConvertor, &position);
#endif
    hipDeviceSynchronize();
    
#if defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    return -99;
#else
    // /* copy stack and description data back to CPU */
    // hipMemcpy(cuda_desc_h, cuda_desc_d, sizeof(ddt_cuda_desc_t), hipMemcpyDeviceToHost);
    //
    // for (i = 0; i < pConvertor->stack_size; i++) {
    //     pConvertor->pStack[i] = cuda_desc_h->pStack[i];
    // }
    //
    // pConvertor->stack_pos = cuda_desc_h->stack_pos;
    // *out_size = cuda_desc_h->out_size;
    // *max_data = cuda_desc_h->max_data;
    // pConvertor->bConverted = cuda_desc_h->bConverted;
    // pConvertor->local_size = cuda_desc_h->local_size;
    //
    // for (i = 0; i < *out_size; i++) {
    //     iov[i].iov_len = cuda_desc_h->iov[i].iov_len;
    // }
    //
    if( pConvertor->bConverted == pConvertor->local_size ) {
        // pConvertor->flags |= CONVERTOR_COMPLETED;
        return 1;
    }

    return 0;
#endif /* OPAL_DATATYPE_CUDA_DRY_RUN */
                                                  
}

void pack_contiguous_loop_cuda( dt_elem_desc_t* ELEM,
                                uint32_t* COUNT,
                                unsigned char** SOURCE,
                                unsigned char** DESTINATION,
                                size_t* SPACE )
{
    ddt_loop_desc_t *_loop = (ddt_loop_desc_t*)(ELEM);
    ddt_endloop_desc_t* _end_loop = (ddt_endloop_desc_t*)((ELEM) + _loop->items);
    unsigned char* _source = (*SOURCE) + _end_loop->first_elem_disp;
    uint32_t _copy_loops = *(COUNT);
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _destination = *(DESTINATION);

    printf("I am in pack_contiguous_loop_cuda\n");

    if( (_copy_loops * _end_loop->size) > *(SPACE) )
        _copy_loops = (uint32_t)(*(SPACE) / _end_loop->size);

#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    _source = pBaseBuf_GPU;
    _destination = (unsigned char*)cuda_desc_h->iov[0].iov_base;
#endif
    
    tasks_per_block = THREAD_PER_BLOCK * TASK_PER_THREAD;
    num_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;
    pack_contiguous_loop_cuda_kernel_global<<<192, 4*THREAD_PER_BLOCK>>>(_copy_loops, _end_loop->size, _loop->extent, _source, _destination);

#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)    
    *(SOURCE) = _source +  _loop->extent*_copy_loops - _end_loop->first_elem_disp;
    *(DESTINATION) = *(DESTINATION)  + _copy_loops * _end_loop->size;
    *(SPACE) -= _copy_loops * _end_loop->size;
    *(COUNT) -= _copy_loops;
#endif
    
    hipDeviceSynchronize();
}


void pack_predefined_data_cuda( dt_elem_desc_t* ELEM,
                                uint32_t* COUNT,
                                unsigned char** SOURCE,
                                unsigned char** DESTINATION,
                                size_t* SPACE )
{
    uint32_t _copy_count = *(COUNT);
    size_t _copy_blength;
    ddt_elem_desc_t* _elem = &((ELEM)->elem);
    unsigned char* _source = (*SOURCE) + _elem->disp;
    uint32_t num_blocks, tasks_per_block;
    unsigned char* _destination = *(DESTINATION);

    _copy_blength = 8;//opal_datatype_basicDatatypes[_elem->common.type]->size;
    if( (_copy_count * _copy_blength) > *(SPACE) ) {
        _copy_count = (uint32_t)(*(SPACE) / _copy_blength);
        if( 0 == _copy_count ) return;  /* nothing to do */
    }
    
#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)
    _source = pBaseBuf_GPU;
    _destination = (unsigned char*)cuda_desc_h->iov[0].iov_base;
#endif
    
    tasks_per_block = THREAD_PER_BLOCK*4;
    num_blocks = (*COUNT + tasks_per_block - 1) / tasks_per_block;

    DBGPRINT("num_blocks %d, thread %d\n", num_blocks, tasks_per_block);
    DBGPRINT( "GPU pack 1. memcpy( %p, %p, %lu ) => space %lu\n", _destination, _source, (unsigned long)_copy_count, (unsigned long)(*(SPACE)) );
    
    pack_contiguous_loop_cuda_kernel_global<<<1, THREAD_PER_BLOCK, 0, cuda_streams->opal_cuda_stream[cuda_streams->current_stream_id]>>>(_copy_count, _copy_blength, _elem->extent, _source, _destination);
    cuda_streams->current_stream_id ++;
    cuda_streams->current_stream_id = cuda_streams->current_stream_id % NB_STREAMS;
    
#if !defined(OPAL_DATATYPE_CUDA_DRY_RUN)  
    _copy_blength *= _copy_count;
    *(SOURCE)  = _source + _elem->extent*_copy_count - _elem->disp;
    *(DESTINATION) += _copy_blength;
    *(SPACE)  -= _copy_blength;
    *(COUNT)  -= _copy_count;
#endif
    
    pBaseBuf_GPU += _elem->extent*_copy_count;
    cuda_desc_h->iov[0].iov_base = (unsigned char*)cuda_desc_h->iov[0].iov_base + _copy_blength;
 //   hipDeviceSynchronize();
}

